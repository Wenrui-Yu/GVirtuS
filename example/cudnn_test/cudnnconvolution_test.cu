#include <iostream>
#include <hipDNN.h>
#include <hip/hip_runtime.h>
#include "cudnn_utils.h"

int main() {
    hipdnnHandle_t cudnn;
    checkCUDNN(hipdnnCreate(&cudnn));

    int n = 1, c = 1, h = 5, w = 5;
    int k = 1, r = 3, s = 3; // Filter: KCRS = 1,1,3,3

    hipdnnTensorDescriptor_t dyDesc, dxDesc;
    hipdnnFilterDescriptor_t wDesc;
    hipdnnConvolutionDescriptor_t convDesc;

    checkCUDNN(hipdnnCreateTensorDescriptor(&dyDesc));
    checkCUDNN(hipdnnCreateTensorDescriptor(&dxDesc));
    checkCUDNN(hipdnnCreateFilterDescriptor(&wDesc));
    checkCUDNN(hipdnnCreateConvolutionDescriptor(&convDesc));

    checkCUDNN(hipdnnSetTensor4dDescriptor(dyDesc, HIPDNN_TENSOR_NCHW, HIPDNN_DATA_FLOAT, n, k, h-2, w-2));
    checkCUDNN(hipdnnSetTensor4dDescriptor(dxDesc, HIPDNN_TENSOR_NCHW, HIPDNN_DATA_FLOAT, n, c, h, w));

    checkCUDNN(hipdnnSetFilter4dDescriptor(wDesc, HIPDNN_DATA_FLOAT, HIPDNN_TENSOR_NCHW, k, c, r, s));

    checkCUDNN(hipdnnSetConvolution2dDescriptor(
        convDesc, 0, 0, 1, 1, 1, 1, HIPDNN_CROSS_CORRELATION, HIPDNN_DATA_FLOAT));

    size_t workspace_bytes = 0;
    checkCUDNN(hipdnnGetConvolutionBackwardDataWorkspaceSize(
        cudnn, wDesc, dyDesc, convDesc, dxDesc, HIPDNN_CONVOLUTION_BWD_DATA_ALGO_0, &workspace_bytes));

    std::cout << "Workspace size: " << workspace_bytes << " bytes" << std::endl;

    void* d_workspace = nullptr;
    if (workspace_bytes > 0)
        checkCUDA(hipMalloc(&d_workspace, workspace_bytes));

    float h_dy[9] = {1,2,3,4,5,6,7,8,9}; // 3x3
    float h_w[9]  = {1,1,1,1,1,1,1,1,1}; // 3x3
    float h_dx[25] = {0}; // 5x5

    float *d_dy, *d_w, *d_dx;
    checkCUDA(hipMalloc(&d_dy, sizeof(h_dy)));
    checkCUDA(hipMalloc(&d_w, sizeof(h_w)));
    checkCUDA(hipMalloc(&d_dx, sizeof(h_dx)));

    checkCUDA(hipMemcpy(d_dy, h_dy, sizeof(h_dy), hipMemcpyHostToDevice));
    checkCUDA(hipMemcpy(d_w, h_w, sizeof(h_w), hipMemcpyHostToDevice));

    float alpha = 1.0f, beta = 0.0f;
    checkCUDNN(hipdnnConvolutionBackwardData(
        cudnn, &alpha, wDesc, d_w, dyDesc, d_dy, convDesc, HIPDNN_CONVOLUTION_BWD_DATA_ALGO_0,
        d_workspace, workspace_bytes, &beta, dxDesc, d_dx));

    checkCUDA(hipMemcpy(h_dx, d_dx, sizeof(h_dx), hipMemcpyDeviceToHost));

    std::cout << "Backward Data result (dx):" << std::endl;
    for (int i = 0; i < 25; ++i) {
        std::cout << h_dx[i] << " ";
        if ((i+1)%5 == 0) std::cout << std::endl;
    }

    if (workspace_bytes > 0) hipFree(d_workspace);
    hipFree(d_dy);
    hipFree(d_w);
    hipFree(d_dx);

    hipdnnDestroyTensorDescriptor(dyDesc);
    hipdnnDestroyTensorDescriptor(dxDesc);
    hipdnnDestroyFilterDescriptor(wDesc);
    hipdnnDestroyConvolutionDescriptor(convDesc);
    hipdnnDestroy(cudnn);

    return 0;
}
