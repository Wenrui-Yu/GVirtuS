#include <iostream>
#include <hipDNN.h>
#include <hip/hip_runtime.h>
#include "cudnn_utils.h"

int main() {
    hipdnnHandle_t cudnn;
    checkCUDNN(hipdnnCreate(&cudnn));

    hipdnnLRNDescriptor_t lrnDesc;
    checkCUDNN(hipdnnCreateLRNDescriptor(&lrnDesc));
    checkCUDNN(hipdnnSetLRNDescriptor(lrnDesc, 5, 1e-4, 0.75, 2.0));

    hipdnnTensorDescriptor_t tensorDesc;
    checkCUDNN(hipdnnCreateTensorDescriptor(&tensorDesc));
    checkCUDNN(hipdnnSetTensor4dDescriptor(tensorDesc, HIPDNN_TENSOR_NCHW, HIPDNN_DATA_FLOAT, 1, 1, 3, 3));

    float h_input[9] = {
        1.0f, 2.0f, 3.0f,
        4.0f, 5.0f, 6.0f,
        7.0f, 8.0f, 9.0f
    };
    float h_output[9] = {0};

    float *d_input, *d_output;
    checkCUDA(hipMalloc(&d_input, sizeof(h_input)));
    checkCUDA(hipMalloc(&d_output, sizeof(h_output)));
    checkCUDA(hipMemcpy(d_input, h_input, sizeof(h_input), hipMemcpyHostToDevice));

    float alpha = 1.0f, beta = 0.0f;

    checkCUDNN(hipdnnLRNCrossChannelForward(
        cudnn,
        lrnDesc,
        HIPDNN_LRN_CROSS_CHANNEL,
        &alpha,
        tensorDesc,
        d_input,
        &beta,
        tensorDesc,
        d_output
    ));

    checkCUDA(hipMemcpy(h_output, d_output, sizeof(h_output), hipMemcpyDeviceToHost));

    std::cout << "LRN output:" << std::endl;
    for (int i = 0; i < 9; ++i) {
        std::cout << h_output[i] << " ";
        if ((i + 1) % 3 == 0) std::cout << std::endl;
    }

    checkCUDA(hipFree(d_input));
    checkCUDA(hipFree(d_output));
    checkCUDNN(hipdnnDestroyTensorDescriptor(tensorDesc));
    checkCUDNN(hipdnnDestroyLRNDescriptor(lrnDesc));
    checkCUDNN(hipdnnDestroy(cudnn));

    return 0;
}
