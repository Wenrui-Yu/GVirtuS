#include "hip/hip_runtime.h"
#include <iostream>
#include <hipDNN.h>
#include <hip/hip_runtime.h>
#include "cudnn_utils.h"
template<typename T>
void memcpyChunked(T* d_dst, const T* h_src, size_t count)
{
    const size_t CHUNK = 1 << 18;          // 256 Ki 元素 ≈ 1 MiB（float）
    for (size_t off = 0; off < count; off += CHUNK) {
        size_t cur = std::min(CHUNK, count - off);
        checkCUDA(hipMemcpy(d_dst + off,
                              h_src + off,
                              cur * sizeof(T),
                              hipMemcpyHostToDevice));
    }
}

int main() {
    try {
        hipdnnHandle_t cudnn;
        checkCUDNN(hipdnnCreate(&cudnn));
        hipdnnPoolingDescriptor_t poolingDesc;
        checkCUDNN(hipdnnCreatePoolingDescriptor(&poolingDesc));
        checkCUDNN(hipdnnSetPooling2dDescriptor(
            poolingDesc,
            HIPDNN_POOLING_MAX,
            HIPDNN_PROPAGATE_NAN,
            2, 2,   // windowHeight, windowWidth
            0, 0,   // verticalPadding, horizontalPadding
            2, 2)); // verticalStride, horizontalStride

        hipdnnTensorDescriptor_t inputDesc, outputDesc;
        checkCUDNN(hipdnnCreateTensorDescriptor(&inputDesc));
        checkCUDNN(hipdnnCreateTensorDescriptor(&outputDesc));

        checkCUDNN(hipdnnSetTensor4dDescriptor(
            inputDesc,
            HIPDNN_TENSOR_NCHW,
            HIPDNN_DATA_FLOAT,
            1, 1, 4, 4));

        checkCUDNN(hipdnnSetTensor4dDescriptor(
            outputDesc,
            HIPDNN_TENSOR_NCHW,
            HIPDNN_DATA_FLOAT,
            1, 1, 2, 2));

        float h_input[16] = {
            1, 2, 3, 4,
            5, 6, 7, 8,
            9,10,11,12,
           13,14,15,16
        };
        float h_output[4] = {0};
        int N=16;
        float *d_input, *d_output;
        checkCUDA(hipMalloc(&d_input, sizeof(h_input)));
        checkCUDA(hipMalloc(&d_output, sizeof(h_output)));

        memcpyChunked<float>(d_input, h_input, N);

        // 如果你想继续沿用宏来捕获最后一次错误（可选）
        checkCUDA(hipDeviceSynchronize());

        float alpha = 1.0f, beta = 0.0f;
        checkCUDNN(hipdnnPoolingForward(
            cudnn,
            poolingDesc,
            &alpha,
            inputDesc,
            d_input,
            &beta,
            outputDesc,
            d_output));

        checkCUDA(hipMemcpy(h_output, d_output, sizeof(h_output), hipMemcpyDeviceToHost));

        std::cout << "Pooling output:" << std::endl;
        for (int i = 0; i < 4; ++i) {
            std::cout << h_output[i] << " ";
        }
        std::cout << std::endl;

        hipFree(d_input);
        hipFree(d_output);
        hipdnnDestroyTensorDescriptor(inputDesc);
        hipdnnDestroyTensorDescriptor(outputDesc);
        hipdnnDestroyPoolingDescriptor(poolingDesc);
        hipdnnDestroy(cudnn);
    }
    catch (const std::exception& e) {
        std::cerr << "Caught std::exception: " << e.what() << std::endl;
        return EXIT_FAILURE;
    }
    catch (const std::string& s) {
        std::cerr << "Caught std::string exception: " << s << std::endl;
        return EXIT_FAILURE;
    }
    catch (const char* msg) {
        std::cerr << "Caught C-string exception: " << msg << std::endl;
        return EXIT_FAILURE;
    }
    catch (...) {
        std::cerr << "Caught unknown exception." << std::endl;
        return EXIT_FAILURE;
    }

    return 0;
}
