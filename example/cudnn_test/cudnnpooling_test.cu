#include <iostream>
#include <hipDNN.h>
#include <hip/hip_runtime.h>
#include "cudnn_utils.h"

int main() {
    try {
        hipdnnHandle_t cudnn;
        checkCUDNN(hipdnnCreate(&cudnn));
        hipdnnPoolingDescriptor_t poolingDesc;
        checkCUDNN(hipdnnCreatePoolingDescriptor(&poolingDesc));
        checkCUDNN(hipdnnSetPooling2dDescriptor(
            poolingDesc,
            HIPDNN_POOLING_MAX,
            HIPDNN_PROPAGATE_NAN,
            2, 2,   // windowHeight, windowWidth
            0, 0,   // verticalPadding, horizontalPadding
            2, 2)); // verticalStride, horizontalStride

        hipdnnTensorDescriptor_t inputDesc, outputDesc;
        checkCUDNN(hipdnnCreateTensorDescriptor(&inputDesc));
        checkCUDNN(hipdnnCreateTensorDescriptor(&outputDesc));

        checkCUDNN(hipdnnSetTensor4dDescriptor(
            inputDesc,
            HIPDNN_TENSOR_NCHW,
            HIPDNN_DATA_FLOAT,
            1, 1, 4, 4));

        checkCUDNN(hipdnnSetTensor4dDescriptor(
            outputDesc,
            HIPDNN_TENSOR_NCHW,
            HIPDNN_DATA_FLOAT,
            1, 1, 2, 2));

        float h_input[16] = {
            1, 2, 3, 4,
            5, 6, 7, 8,
            9,10,11,12,
           13,14,15,16
        };
        float h_output[4] = {0};

        float *d_input, *d_output;
        checkCUDA(hipMalloc(&d_input, sizeof(h_input)));
        checkCUDA(hipMalloc(&d_output, sizeof(h_output)));

        checkCUDA(hipMemcpy(d_input, h_input, sizeof(h_input), hipMemcpyHostToDevice));

        float alpha = 1.0f, beta = 0.0f;
        checkCUDNN(hipdnnPoolingForward(
            cudnn,
            poolingDesc,
            &alpha,
            inputDesc,
            d_input,
            &beta,
            outputDesc,
            d_output));

        checkCUDA(hipMemcpy(h_output, d_output, sizeof(h_output), hipMemcpyDeviceToHost));

        std::cout << "Pooling output:" << std::endl;
        for (int i = 0; i < 4; ++i) {
            std::cout << h_output[i] << " ";
        }
        std::cout << std::endl;

        hipFree(d_input);
        hipFree(d_output);
        hipdnnDestroyTensorDescriptor(inputDesc);
        hipdnnDestroyTensorDescriptor(outputDesc);
        hipdnnDestroyPoolingDescriptor(poolingDesc);
        hipdnnDestroy(cudnn);
    }
    catch (const std::exception& e) {
        std::cerr << "Caught std::exception: " << e.what() << std::endl;
        return EXIT_FAILURE;
    }
    catch (const std::string& s) {
        std::cerr << "Caught std::string exception: " << s << std::endl;
        return EXIT_FAILURE;
    }
    catch (const char* msg) {
        std::cerr << "Caught C-string exception: " << msg << std::endl;
        return EXIT_FAILURE;
    }
    catch (...) {
        std::cerr << "Caught unknown exception." << std::endl;
        return EXIT_FAILURE;
    }

    return 0;
}
