#include <iostream>
#include <hipDNN.h>
#include <hip/hip_runtime.h>
#include "cudnn_utils.h"
#ifndef CUDNN_TENSOR_TRANSFORM_IDENTITY
#define CUDNN_TENSOR_TRANSFORM_IDENTITY 0
#endif


void printTensorNdDescriptor(hipdnnTensorDescriptor_t desc) {
    int nbDims = 0;
    hipdnnDataType_t dataType;
    int dims[10] = {0};
    int strides[10] = {0};
    checkCUDNN(hipdnnGetTensorNdDescriptor(desc, 10, &dataType, &nbDims, dims, strides));
    std::cout << "TensorNdDescriptor info:\n";
    std::cout << "Number of dims: " << nbDims << "\n";
    std::cout << "Data type: " << dataType << "\n";
    std::cout << "Dims: ";
    for (int i = 0; i < nbDims; ++i) std::cout << dims[i] << " ";
    std::cout << "\nStrides: ";
    for (int i = 0; i < nbDims; ++i) std::cout << strides[i] << " ";
    std::cout << std::endl;
}

int main() {
    hipdnnHandle_t cudnn;
    checkCUDNN(hipdnnCreate(&cudnn));

    hipdnnTensorDescriptor_t inputDesc;
    checkCUDNN(hipdnnCreateTensorDescriptor(&inputDesc));
    checkCUDNN(hipdnnSetTensor4dDescriptor(inputDesc, HIPDNN_TENSOR_NCHW, HIPDNN_DATA_FLOAT, 1, 1, 2, 3));

    hipdnnTensorDescriptor_t outputDesc;
    checkCUDNN(hipdnnCreateTensorDescriptor(&outputDesc));
    checkCUDNN(hipdnnSetTensor4dDescriptor(outputDesc, HIPDNN_TENSOR_NCHW, HIPDNN_DATA_FLOAT, 1, 1, 3, 2));

    cudnnTensorTransformDescriptor_t transformDesc;
    checkCUDNN(cudnnCreateTensorTransformDescriptor(&transformDesc));
    
    int nbDims = 4;
int dims[4]        = {1, 1, 2, 3};
int permuteDims[4] = {0, 1, 2, 3};
int offsets[4]     = {0, 0, 0, 0};
int strides[4]     = {6, 6, 3, 1};
    checkCUDNN(cudnnSetTensorTransformDescriptor(
    transformDesc,
    CUDNN_TENSOR_TRANSFORM_IDENTITY,
    nbDims,
    dims,
    permuteDims,
    offsets,
    strides));

    float h_input[6] = {1, 2, 3, 4, 5, 6};
    float h_output[6] = {0};

    float* d_input;
    float* d_output;
    checkCUDA(hipMalloc(&d_input, sizeof(h_input)));
    checkCUDA(hipMalloc(&d_output, sizeof(h_output)));

    checkCUDA(hipMemcpy(d_input, h_input, sizeof(h_input), hipMemcpyHostToDevice));

    checkCUDNN(cudnnTransformTensorEx(
        cudnn,
        transformDesc,
        nullptr,
        inputDesc,
        d_input,
        nullptr,
        outputDesc,
        d_output));

    checkCUDA(hipMemcpy(h_output, d_output, sizeof(h_output), hipMemcpyDeviceToHost));

    std::cout << "Output of cudnnTransformTensorEx:\n";
    for (int i = 0; i < 6; ++i) {
        std::cout << h_output[i] << " ";
        if ((i + 1) % 2 == 0) std::cout << "\n";
    }

    std::cout << "\nInput Tensor Descriptor:\n";
    printTensorNdDescriptor(inputDesc);

    std::cout << "\nOutput Tensor Descriptor:\n";
    printTensorNdDescriptor(outputDesc);

    checkCUDA(hipFree(d_input));
    checkCUDA(hipFree(d_output));
    checkCUDNN(cudnnDestroyTensorTransformDescriptor(transformDesc));
    checkCUDNN(hipdnnDestroyTensorDescriptor(inputDesc));
    checkCUDNN(hipdnnDestroyTensorDescriptor(outputDesc));
    checkCUDNN(hipdnnDestroy(cudnn));

    return 0;
}
