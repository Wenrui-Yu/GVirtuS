#include <gtest/gtest.h>
#include <hip/hip_runtime.h>
#include <hipDNN.h>

#define CUDA_CHECK(err) ASSERT_EQ((err), hipSuccess)
#define CUDNN_CHECK(err) ASSERT_EQ((err), HIPDNN_STATUS_SUCCESS)

TEST(cuDNN, CreateDestroy) {
    hipdnnHandle_t handle;
    CUDNN_CHECK(hipdnnCreate(&handle));
    CUDNN_CHECK(hipdnnDestroy(handle));
}

TEST(cuDNN, SetStreamDestroy) {
    hipdnnHandle_t handle;
    hipStream_t stream;
    CUDNN_CHECK(hipdnnCreate(&handle));
    CUDA_CHECK(hipStreamCreate(&stream));
    CUDNN_CHECK(hipdnnSetStream(handle, stream));
    CUDA_CHECK(hipStreamDestroy(stream));
    CUDNN_CHECK(hipdnnDestroy(handle));
}

TEST(cuDNN, AddTensor) {
    hipdnnHandle_t handle;
    CUDNN_CHECK(hipdnnCreate(&handle));

    const int N = 1, C = 1, H = 2, W = 2;
    const int size = N * C * H * W;
    float h_A[] = {1, 2, 3, 4};
    float h_B[] = {10, 20, 30, 40};

    float *d_A, *d_B;
    CUDA_CHECK(hipMalloc(&d_A, size * sizeof(float)));
    CUDA_CHECK(hipMalloc(&d_B, size * sizeof(float)));

    CUDA_CHECK(hipMemcpy(d_A, h_A, sizeof(h_A), hipMemcpyHostToDevice));
    CUDA_CHECK(hipMemcpy(d_B, h_B, sizeof(h_B), hipMemcpyHostToDevice));

    hipdnnTensorDescriptor_t desc;
    CUDNN_CHECK(hipdnnCreateTensorDescriptor(&desc));
    CUDNN_CHECK(hipdnnSetTensor4dDescriptor(desc, 
                                           HIPDNN_TENSOR_NCHW, 
                                           HIPDNN_DATA_FLOAT, 
                                           N, C, H, W));

    float alpha = 1.0f;
    float beta  = 1.0f;

    // B = alpha * A + beta * B
    CUDNN_CHECK(hipdnnAddTensor(handle,
                               &alpha,
                               desc, d_A,
                               &beta,
                               desc, d_B));

    float h_result[size];
    CUDA_CHECK(hipMemcpy(h_result, d_B, sizeof(h_result), hipMemcpyDeviceToHost));

    // Expected result: B[i] = A[i] + B[i]
    EXPECT_FLOAT_EQ(h_result[0], 11.0f);
    EXPECT_FLOAT_EQ(h_result[1], 22.0f);
    EXPECT_FLOAT_EQ(h_result[2], 33.0f);
    EXPECT_FLOAT_EQ(h_result[3], 44.0f);

    CUDNN_CHECK(hipdnnDestroyTensorDescriptor(desc));
    CUDA_CHECK(hipFree(d_A));
    CUDA_CHECK(hipFree(d_B));
    CUDNN_CHECK(hipdnnDestroy(handle));
}
