#include "hip/hip_runtime.h"
#include <gtest/gtest.h>
#include <hip/hip_runtime.h>
#include <hipDNN.h>
#include "cudnn_utils.h"

//#define CUDA_CHECK(err) ASSERT_EQ((err), hipSuccess)
//#define CUDNN_CHECK(err) ASSERT_EQ((err), HIPDNN_STATUS_SUCCESS)
#define CUDA_CHECK(err) checkCUDA(err)
#define CUDNN_CHECK(err) checkCUDNN(err)

class CuDNNTestWithCatch : public ::testing::Test {
protected:
    void RunWithExceptionHandling(std::function<void()> testFunc) {
        try {
            testFunc();
        }
        catch (const std::exception& e) {
            std::cerr << "Caught std::exception: " << e.what() << std::endl;
            FAIL() << "Test failed due to std::exception";
        }
        catch (const std::string& s) {
            std::cerr << "Caught std::string exception: " << s << std::endl;
            FAIL() << "Test failed due to std::string exception";
        }
        catch (const char* msg) {
            std::cerr << "Caught C-string exception: " << msg << std::endl;
            FAIL() << "Test failed due to C-string exception";
        }
        catch (...) {
            std::cerr << "Caught unknown exception." << std::endl;
            FAIL() << "Test failed due to unknown exception";
        }
    }
};

TEST_F(CuDNNTestWithCatch, CreateDestroy) {
    RunWithExceptionHandling([](){
    hipdnnHandle_t handle;
    CUDNN_CHECK(hipdnnCreate(&handle));
    CUDNN_CHECK(hipdnnDestroy(handle));
    });
}

TEST_F(CuDNNTestWithCatch, GetVersion) {
    RunWithExceptionHandling([](){
    size_t version = hipdnnGetVersion();
    ASSERT_GT(version, 0);
    std::cout << "cuDNN version: " << version << std::endl;
    });
}

TEST_F(CuDNNTestWithCatch, SetGetStream) {
    RunWithExceptionHandling([](){
    hipdnnHandle_t handle;
    CUDNN_CHECK(hipdnnCreate(&handle));

    hipStream_t stream;
    CUDA_CHECK(hipStreamCreate(&stream));

    CUDNN_CHECK(hipdnnSetStream(handle, stream));

    hipStream_t got_stream;
    CUDNN_CHECK(hipdnnGetStream(handle, &got_stream));
    ASSERT_EQ(stream, got_stream);

    CUDA_CHECK(hipStreamDestroy(stream));
    CUDNN_CHECK(hipdnnDestroy(handle));
    });
}

TEST_F(CuDNNTestWithCatch, TensorDescriptorCreateDestroy) {
    RunWithExceptionHandling([](){
    hipdnnTensorDescriptor_t desc;
    CUDNN_CHECK(hipdnnCreateTensorDescriptor(&desc));
    CUDNN_CHECK(hipdnnDestroyTensorDescriptor(desc));
    });
}

TEST_F(CuDNNTestWithCatch, TensorDescriptorSetGet) {
    RunWithExceptionHandling([](){
    hipdnnTensorDescriptor_t desc;
    CUDNN_CHECK(hipdnnCreateTensorDescriptor(&desc));

    CUDNN_CHECK(hipdnnSetTensor4dDescriptor(desc, HIPDNN_TENSOR_NCHW, HIPDNN_DATA_FLOAT, 1, 2, 3, 4));

    hipdnnDataType_t dataType;
    int n, c, h, w, nStride, cStride, hStride, wStride;

    CUDNN_CHECK(hipdnnGetTensor4dDescriptor(desc, &dataType, &n, &c, &h, &w, &nStride, &cStride, &hStride, &wStride));
    ASSERT_EQ(n, 1);
    ASSERT_EQ(c, 2);
    ASSERT_EQ(h, 3);
    ASSERT_EQ(w, 4);

    CUDNN_CHECK(hipdnnDestroyTensorDescriptor(desc));
    });
}

TEST_F(CuDNNTestWithCatch, ActivationForwardReLU) {
    RunWithExceptionHandling([](){
    hipdnnHandle_t handle;
    CUDNN_CHECK(hipdnnCreate(&handle));

    hipdnnTensorDescriptor_t desc;
    CUDNN_CHECK(hipdnnCreateTensorDescriptor(&desc));
    CUDNN_CHECK(hipdnnSetTensor4dDescriptor(desc, HIPDNN_TENSOR_NCHW, HIPDNN_DATA_FLOAT, 1, 1, 1, 3));

    hipdnnActivationDescriptor_t actDesc;
    CUDNN_CHECK(hipdnnCreateActivationDescriptor(&actDesc));
    CUDNN_CHECK(hipdnnSetActivationDescriptor(actDesc, HIPDNN_ACTIVATION_RELU, HIPDNN_PROPAGATE_NAN, 0.0));

    float h_input[] = {-1.0f, 0.0f, 2.0f};
    float h_output[3] = {};

    float *d_input, *d_output;
    CUDA_CHECK(hipMalloc(&d_input, sizeof(h_input)));
    CUDA_CHECK(hipMalloc(&d_output, sizeof(h_output)));
    CUDA_CHECK(hipMemcpy(d_input, h_input, sizeof(h_input), hipMemcpyHostToDevice));

    float alpha = 1.0f, beta = 0.0f;
    CUDNN_CHECK(hipdnnActivationForward(handle, actDesc, &alpha, desc, d_input, &beta, desc, d_output));
    CUDA_CHECK(hipMemcpy(h_output, d_output, sizeof(h_output), hipMemcpyDeviceToHost));

    ASSERT_FLOAT_EQ(h_output[0], 0.0f);
    ASSERT_FLOAT_EQ(h_output[1], 0.0f);
    ASSERT_FLOAT_EQ(h_output[2], 2.0f);

    CUDA_CHECK(hipFree(d_input));
    CUDA_CHECK(hipFree(d_output));
    CUDNN_CHECK(hipdnnDestroyTensorDescriptor(desc));
    CUDNN_CHECK(hipdnnDestroyActivationDescriptor(actDesc));
    CUDNN_CHECK(hipdnnDestroy(handle));
    });
}

TEST_F(CuDNNTestWithCatch, PoolingForwardMax) {
    RunWithExceptionHandling([](){
    hipdnnHandle_t handle;
    CUDNN_CHECK(hipdnnCreate(&handle));

    hipdnnTensorDescriptor_t in_desc, out_desc;
    hipdnnPoolingDescriptor_t pool_desc;

    CUDNN_CHECK(hipdnnCreateTensorDescriptor(&in_desc));
    CUDNN_CHECK(hipdnnCreateTensorDescriptor(&out_desc));
    CUDNN_CHECK(hipdnnCreatePoolingDescriptor(&pool_desc));

    CUDNN_CHECK(hipdnnSetTensor4dDescriptor(in_desc, HIPDNN_TENSOR_NCHW, HIPDNN_DATA_FLOAT, 1, 1, 2, 2));
    CUDNN_CHECK(hipdnnSetTensor4dDescriptor(out_desc, HIPDNN_TENSOR_NCHW, HIPDNN_DATA_FLOAT, 1, 1, 1, 1));
    CUDNN_CHECK(hipdnnSetPooling2dDescriptor(pool_desc, HIPDNN_POOLING_MAX, HIPDNN_PROPAGATE_NAN,
                                            2, 2, 0, 0, 1, 1));

    float h_input[] = {1, 2, 3, 4}; // max is 4
    float h_output[1] = {};

    float *d_input, *d_output;
    CUDA_CHECK(hipMalloc(&d_input, sizeof(h_input)));
    CUDA_CHECK(hipMalloc(&d_output, sizeof(h_output)));
    CUDA_CHECK(hipMemcpy(d_input, h_input, sizeof(h_input), hipMemcpyHostToDevice));

    float alpha = 1.0f, beta = 0.0f;
    CUDNN_CHECK(hipdnnPoolingForward(handle, pool_desc, &alpha, in_desc, d_input, &beta, out_desc, d_output));
    CUDA_CHECK(hipMemcpy(h_output, d_output, sizeof(h_output), hipMemcpyDeviceToHost));

    ASSERT_FLOAT_EQ(h_output[0], 4.0f);

    CUDA_CHECK(hipFree(d_input));
    CUDA_CHECK(hipFree(d_output));
    CUDNN_CHECK(hipdnnDestroyTensorDescriptor(in_desc));
    CUDNN_CHECK(hipdnnDestroyTensorDescriptor(out_desc));
    CUDNN_CHECK(hipdnnDestroyPoolingDescriptor(pool_desc));
    CUDNN_CHECK(hipdnnDestroy(handle));
    });
}

TEST_F(CuDNNTestWithCatch, FilterDescriptorCreateDestroy) {
    RunWithExceptionHandling([](){
    hipdnnFilterDescriptor_t filterDesc;
    CUDNN_CHECK(hipdnnCreateFilterDescriptor(&filterDesc));
    CUDNN_CHECK(hipdnnDestroyFilterDescriptor(filterDesc));
    });
}

TEST_F(CuDNNTestWithCatch, ErrorString) {
    RunWithExceptionHandling([](){
    const char* msg = hipdnnGetErrorString(HIPDNN_STATUS_ALLOC_FAILED);
    ASSERT_TRUE(msg != nullptr);
    });
}

TEST_F(CuDNNTestWithCatch, AddTensor) {
    RunWithExceptionHandling([](){
        hipdnnHandle_t handle;
        CUDNN_CHECK(hipdnnCreate(&handle));

        const int N = 1, C = 1, H = 2, W = 2;
        const int size = N * C * H * W;
        float h_A[] = {1, 2, 3, 4};
        float h_B[] = {10, 20, 30, 40};

        float *d_A, *d_B;
        CUDA_CHECK(hipMalloc(&d_A, size * sizeof(float)));
        CUDA_CHECK(hipMalloc(&d_B, size * sizeof(float)));

        CUDA_CHECK(hipMemcpy(d_A, h_A, sizeof(h_A), hipMemcpyHostToDevice));
        CUDA_CHECK(hipMemcpy(d_B, h_B, sizeof(h_B), hipMemcpyHostToDevice));

        hipdnnTensorDescriptor_t desc;
        CUDNN_CHECK(hipdnnCreateTensorDescriptor(&desc));
        CUDNN_CHECK(hipdnnSetTensor4dDescriptor(desc, 
                                               HIPDNN_TENSOR_NCHW, 
                                               HIPDNN_DATA_FLOAT, 
                                               N, C, H, W));

        float alpha = 1.0f;
        float beta  = 1.0f;

        // B = alpha * A + beta * B
        CUDNN_CHECK(hipdnnAddTensor(handle,
                                   &alpha,
                                   desc, d_A,
                                   &beta,
                                   desc, d_B));

        float h_result[size];
        CUDA_CHECK(hipMemcpy(h_result, d_B, sizeof(h_result), hipMemcpyDeviceToHost));

        // Expected result: B[i] = A[i] + B[i]
        EXPECT_FLOAT_EQ(h_result[0], 11.0f);
        EXPECT_FLOAT_EQ(h_result[1], 22.0f);
        EXPECT_FLOAT_EQ(h_result[2], 33.0f);
        EXPECT_FLOAT_EQ(h_result[3], 44.0f);

        // Clean up
        CUDNN_CHECK(hipdnnDestroyTensorDescriptor(desc));
        CUDA_CHECK(hipFree(d_A));
        CUDA_CHECK(hipFree(d_B));
        CUDNN_CHECK(hipdnnDestroy(handle));
    });
}

TEST_F(CuDNNTestWithCatch, PoolingForward) {
    RunWithExceptionHandling([](){
        hipdnnHandle_t handle;
        CUDNN_CHECK(hipdnnCreate(&handle));

        const int N = 1, C = 1, H = 2, W = 2;
        const int size = N * C * H * W;
        float h_input[]  = {1.0f, 2.0f, 3.0f, 4.0f};
        float h_output[1];  // Output will be a single value after 2x2 pooling

        float *d_input, *d_output;
        CUDA_CHECK(hipMalloc(&d_input,  size * sizeof(float)));
        CUDA_CHECK(hipMalloc(&d_output, sizeof(float)));

        CUDA_CHECK(hipMemcpy(d_input, h_input, sizeof(h_input), hipMemcpyHostToDevice));

        hipdnnTensorDescriptor_t inputDesc, outputDesc;
        CUDNN_CHECK(hipdnnCreateTensorDescriptor(&inputDesc));
        CUDNN_CHECK(hipdnnCreateTensorDescriptor(&outputDesc));

        // Set tensor descriptors
        CUDNN_CHECK(hipdnnSetTensor4dDescriptor(inputDesc, HIPDNN_TENSOR_NCHW, HIPDNN_DATA_FLOAT, N, C, H, W));
        CUDNN_CHECK(hipdnnSetTensor4dDescriptor(outputDesc, HIPDNN_TENSOR_NCHW, HIPDNN_DATA_FLOAT, N, C, 1, 1));

        // Create and set pooling descriptor
        hipdnnPoolingDescriptor_t poolingDesc;
        CUDNN_CHECK(hipdnnCreatePoolingDescriptor(&poolingDesc));

        CUDNN_CHECK(hipdnnSetPooling2dDescriptor(poolingDesc,
                                                HIPDNN_POOLING_MAX,
                                                HIPDNN_PROPAGATE_NAN,
                                                2, 2,   // window height, width
                                                0, 0,   // padding height, width
                                                2, 2)); // stride height, width

        float alpha = 1.0f, beta = 0.0f;
        CUDNN_CHECK(hipdnnPoolingForward(handle,
                                        poolingDesc,
                                        &alpha,
                                        inputDesc, d_input,
                                        &beta,
                                        outputDesc, d_output));

        CUDA_CHECK(hipMemcpy(h_output, d_output, sizeof(float), hipMemcpyDeviceToHost));

        // Expected output is max(1, 2, 3, 4) = 4.0f
        EXPECT_FLOAT_EQ(h_output[0], 4.0f);

        // Cleanup
        CUDNN_CHECK(hipdnnDestroyPoolingDescriptor(poolingDesc));
        CUDNN_CHECK(hipdnnDestroyTensorDescriptor(inputDesc));
        CUDNN_CHECK(hipdnnDestroyTensorDescriptor(outputDesc));
        CUDA_CHECK(hipFree(d_input));
        CUDA_CHECK(hipFree(d_output));
        CUDNN_CHECK(hipdnnDestroy(handle));
    });
}

TEST_F(CuDNNTestWithCatch, ConvolutionForward) {
    RunWithExceptionHandling([](){
        hipdnnHandle_t handle;
        CUDNN_CHECK(hipdnnCreate(&handle));

        const int N = 1, C = 1, H = 2, W = 2;
        const int size = N * C * H * W;

        float h_input[]  = {1.0f, 2.0f, 3.0f, 4.0f};
        float h_filter[] = {10.0f};  // 1x1 filter
        float h_output[4];

        float *d_input, *d_filter, *d_output;
        CUDA_CHECK(hipMalloc(&d_input,  size * sizeof(float)));
        CUDA_CHECK(hipMalloc(&d_filter, sizeof(float)));
        CUDA_CHECK(hipMalloc(&d_output, size * sizeof(float)));

        CUDA_CHECK(hipMemcpy(d_input, h_input, sizeof(h_input), hipMemcpyHostToDevice));
        CUDA_CHECK(hipMemcpy(d_filter, h_filter, sizeof(h_filter), hipMemcpyHostToDevice));

        // Create descriptors
        hipdnnTensorDescriptor_t inputDesc, outputDesc;
        hipdnnFilterDescriptor_t filterDesc;
        hipdnnConvolutionDescriptor_t convDesc;

        CUDNN_CHECK(hipdnnCreateTensorDescriptor(&inputDesc));
        CUDNN_CHECK(hipdnnCreateTensorDescriptor(&outputDesc));
        CUDNN_CHECK(hipdnnCreateFilterDescriptor(&filterDesc));
        CUDNN_CHECK(hipdnnCreateConvolutionDescriptor(&convDesc));

        CUDNN_CHECK(hipdnnSetTensor4dDescriptor(inputDesc, HIPDNN_TENSOR_NCHW, HIPDNN_DATA_FLOAT, N, C, H, W));
        CUDNN_CHECK(hipdnnSetTensor4dDescriptor(outputDesc, HIPDNN_TENSOR_NCHW, HIPDNN_DATA_FLOAT, N, C, H, W));
        CUDNN_CHECK(hipdnnSetFilter4dDescriptor(filterDesc, HIPDNN_DATA_FLOAT, HIPDNN_TENSOR_NCHW, 1, 1, 1, 1));

        CUDNN_CHECK(hipdnnSetConvolution2dDescriptor(convDesc, 0, 0, 1, 1, 1, 1, HIPDNN_CROSS_CORRELATION, HIPDNN_DATA_FLOAT));

        // Choose algo
        hipdnnConvolutionFwdAlgo_t algo = HIPDNN_CONVOLUTION_FWD_ALGO_IMPLICIT_PRECOMP_GEMM;

        // Workspace
        size_t workspaceBytes = 0;
        CUDNN_CHECK(hipdnnGetConvolutionForwardWorkspaceSize(handle, inputDesc, filterDesc, convDesc, outputDesc, algo, &workspaceBytes));

        void* d_workspace = nullptr;
        if (workspaceBytes > 0)
            CUDA_CHECK(hipMalloc(&d_workspace, workspaceBytes));

        float alpha = 1.0f, beta = 0.0f;
        CUDNN_CHECK(hipdnnConvolutionForward(handle,
                                            &alpha,
                                            inputDesc, d_input,
                                            filterDesc, d_filter,
                                            convDesc, algo,
                                            d_workspace, workspaceBytes,
                                            &beta,
                                            outputDesc, d_output));

        CUDA_CHECK(hipMemcpy(h_output, d_output, sizeof(h_output), hipMemcpyDeviceToHost));

        // Verify results
        EXPECT_FLOAT_EQ(h_output[0], 10.0f);
        EXPECT_FLOAT_EQ(h_output[1], 20.0f);
        EXPECT_FLOAT_EQ(h_output[2], 30.0f);
        EXPECT_FLOAT_EQ(h_output[3], 40.0f);

        // Cleanup
        if (workspaceBytes > 0)
            CUDA_CHECK(hipFree(d_workspace));

        CUDNN_CHECK(hipdnnDestroyTensorDescriptor(inputDesc));
        CUDNN_CHECK(hipdnnDestroyTensorDescriptor(outputDesc));
        CUDNN_CHECK(hipdnnDestroyFilterDescriptor(filterDesc));
        CUDNN_CHECK(hipdnnDestroyConvolutionDescriptor(convDesc));
        CUDA_CHECK(hipFree(d_input));
        CUDA_CHECK(hipFree(d_filter));
        CUDA_CHECK(hipFree(d_output));
        CUDNN_CHECK(hipdnnDestroy(handle));
    });
}

TEST_F(CuDNNTestWithCatch, FilterDescriptorCreateSetGet) {
    RunWithExceptionHandling([](){
        hipdnnHandle_t handle;
        CUDNN_CHECK(hipdnnCreate(&handle));

        hipdnnFilterDescriptor_t filterDesc;
        CUDNN_CHECK(hipdnnCreateFilterDescriptor(&filterDesc));

        // Set descriptor: format NCHW, 1 output, 1 input, 3x3 kernel
        const int k = 1, c = 1, h = 3, w = 3;
        CUDNN_CHECK(hipdnnSetFilter4dDescriptor(filterDesc, HIPDNN_DATA_FLOAT, HIPDNN_TENSOR_NCHW, k, c, h, w));

        // Retrieve and check descriptor values
        hipdnnDataType_t dataType;
        hipdnnTensorFormat_t format;
        int k_ret, c_ret, h_ret, w_ret;
        CUDNN_CHECK(hipdnnGetFilter4dDescriptor(filterDesc, &dataType, &format, &k_ret, &c_ret, &h_ret, &w_ret));

        EXPECT_EQ(dataType, HIPDNN_DATA_FLOAT);
        EXPECT_EQ(format,  HIPDNN_TENSOR_NCHW);
        EXPECT_EQ(k_ret, k);
        EXPECT_EQ(c_ret, c);
        EXPECT_EQ(h_ret, h);
        EXPECT_EQ(w_ret, w);

        // Cleanup
        CUDNN_CHECK(hipdnnDestroyFilterDescriptor(filterDesc));
        CUDNN_CHECK(hipdnnDestroy(handle));
    });
}
/*
TEST_F(CuDNNTestWithCatch, LRNForward) {
    RunWithExceptionHandling([](){
    hipdnnHandle_t handle;
    CUDNN_CHECK(hipdnnCreate(&handle));

    // Tensor dims: NCHW = 1x1x1x5
    const int N = 1, C = 5, H = 1, W = 1;
    const int size = N * C * H * W;

    float h_input[]  = {1.0f, 2.0f, 3.0f, 4.0f, 5.0f};
    float h_output[size];

    float *d_input, *d_output;
    CUDA_CHECK(hipMalloc(&d_input,  size * sizeof(float)));
    CUDA_CHECK(hipMalloc(&d_output, size * sizeof(float)));

    CUDA_CHECK(hipMemcpy(d_input, h_input, sizeof(h_input), hipMemcpyHostToDevice));

    // Create tensor descriptors
    hipdnnTensorDescriptor_t tensorDesc;
    CUDNN_CHECK(hipdnnCreateTensorDescriptor(&tensorDesc));
    CUDNN_CHECK(hipdnnSetTensor4dDescriptor(tensorDesc,
                                           HIPDNN_TENSOR_NCHW,
                                           HIPDNN_DATA_FLOAT,
                                           N, C, H, W));

    // Create LRN descriptor
    hipdnnLRNDescriptor_t lrnDesc;
    CUDNN_CHECK(hipdnnCreateLRNDescriptor(&lrnDesc));

    // Set LRN parameters: local_size, alpha, beta, k
    const unsigned localSize = 3;
    const double alpha = 1e-4;
    const double beta  = 0.75;
    const double k     = 2.0;

    CUDNN_CHECK(hipdnnSetLRNDescriptor(lrnDesc, localSize, alpha, beta, k));

    float alpha1 = 1.0f, beta1 = 0.0f;
    CUDNN_CHECK(hipdnnLRNCrossChannelForward(handle,
                                            lrnDesc,
                                            HIPDNN_LRN_CROSS_CHANNEL,
                                            &alpha1,
                                            tensorDesc, d_input,
                                            &beta1,
                                            tensorDesc, d_output));

    CUDA_CHECK(hipMemcpy(h_output, d_output, sizeof(h_output), hipMemcpyDeviceToHost));

    // Print results (since exact analytical value is tedious, we can sanity check)
    for (int i = 0; i < size; ++i) {
        printf("LRN output[%d] = %f\n", i, h_output[i]);
    }

    // Basic sanity check: output should be less than or equal to input since normalization happens
    for (int i = 0; i < size; ++i) {
        EXPECT_LE(h_output[i], h_input[i]);
    }

    // Cleanup
    CUDNN_CHECK(hipdnnDestroyLRNDescriptor(lrnDesc));
    CUDNN_CHECK(hipdnnDestroyTensorDescriptor(tensorDesc));
    CUDA_CHECK(hipFree(d_input));
    CUDA_CHECK(hipFree(d_output));
    CUDNN_CHECK(hipdnnDestroy(handle));
    });
}
*/
