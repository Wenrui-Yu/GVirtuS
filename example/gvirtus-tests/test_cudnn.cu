#include "hip/hip_runtime.h"
#include <gtest/gtest.h>
#include <hip/hip_runtime.h>
#include <hipDNN.h>
#include "cudnn_utils.h"

//#define CUDA_CHECK(err) ASSERT_EQ((err), hipSuccess)
//#define CUDNN_CHECK(err) ASSERT_EQ((err), HIPDNN_STATUS_SUCCESS)
#define CUDA_CHECK(err) checkCUDA(err)
#define CUDNN_CHECK(err) checkCUDNN(err)

class CuDNNTestWithCatch : public ::testing::Test {
protected:
    void RunWithExceptionHandling(std::function<void()> testFunc) {
        try {
            testFunc();
        }
        catch (const std::exception& e) {
            std::cerr << "Caught std::exception: " << e.what() << std::endl;
            FAIL() << "Test failed due to std::exception";
        }
        catch (const std::string& s) {
            std::cerr << "Caught std::string exception: " << s << std::endl;
            FAIL() << "Test failed due to std::string exception";
        }
        catch (const char* msg) {
            std::cerr << "Caught C-string exception: " << msg << std::endl;
            FAIL() << "Test failed due to C-string exception";
        }
        catch (...) {
            std::cerr << "Caught unknown exception." << std::endl;
            FAIL() << "Test failed due to unknown exception";
        }
    }
};

TEST_F(CuDNNTestWithCatch, CreateDestroy) {
    RunWithExceptionHandling([](){
        hipdnnHandle_t handle;
        CUDNN_CHECK(hipdnnCreate(&handle));
        CUDNN_CHECK(hipdnnDestroy(handle));
    });
}
/*
TEST_F(CuDNNTestWithCatch, SetStreamDestroy) {
    RunWithExceptionHandling([](){
        hipdnnHandle_t handle;
        hipStream_t stream;
        CUDNN_CHECK(hipdnnCreate(&handle));
        CUDA_CHECK(hipStreamCreate(&stream));
        CUDNN_CHECK(hipdnnSetStream(handle, stream));
        CUDA_CHECK(hipStreamDestroy(stream));
        CUDNN_CHECK(hipdnnDestroy(handle));
    });
}
*/
TEST_F(CuDNNTestWithCatch, AddTensor) {
    RunWithExceptionHandling([](){
        hipdnnHandle_t handle;
        CUDNN_CHECK(hipdnnCreate(&handle));

        const int N = 1, C = 1, H = 2, W = 2;
        const int size = N * C * H * W;
        float h_A[] = {1, 2, 3, 4};
        float h_B[] = {10, 20, 30, 40};

        float *d_A, *d_B;
        CUDA_CHECK(hipMalloc(&d_A, size * sizeof(float)));
        CUDA_CHECK(hipMalloc(&d_B, size * sizeof(float)));

        CUDA_CHECK(hipMemcpy(d_A, h_A, sizeof(h_A), hipMemcpyHostToDevice));
        CUDA_CHECK(hipMemcpy(d_B, h_B, sizeof(h_B), hipMemcpyHostToDevice));

        hipdnnTensorDescriptor_t desc;
        CUDNN_CHECK(hipdnnCreateTensorDescriptor(&desc));
        CUDNN_CHECK(hipdnnSetTensor4dDescriptor(desc, 
                                               HIPDNN_TENSOR_NCHW, 
                                               HIPDNN_DATA_FLOAT, 
                                               N, C, H, W));

        float alpha = 1.0f;
        float beta  = 1.0f;

        // B = alpha * A + beta * B
        CUDNN_CHECK(hipdnnAddTensor(handle,
                                   &alpha,
                                   desc, d_A,
                                   &beta,
                                   desc, d_B));

        float h_result[size];
        CUDA_CHECK(hipMemcpy(h_result, d_B, sizeof(h_result), hipMemcpyDeviceToHost));

        // Expected result: B[i] = A[i] + B[i]
        EXPECT_FLOAT_EQ(h_result[0], 11.0f);
        EXPECT_FLOAT_EQ(h_result[1], 22.0f);
        EXPECT_FLOAT_EQ(h_result[2], 33.0f);
        EXPECT_FLOAT_EQ(h_result[3], 44.0f);

        // Clean up
        CUDNN_CHECK(hipdnnDestroyTensorDescriptor(desc));
        CUDA_CHECK(hipFree(d_A));
        CUDA_CHECK(hipFree(d_B));
        CUDNN_CHECK(hipdnnDestroy(handle));
    });
}

TEST_F(CuDNNTestWithCatch, PoolingForward) {
    RunWithExceptionHandling([](){
        hipdnnHandle_t handle;
        CUDNN_CHECK(hipdnnCreate(&handle));

        const int N = 1, C = 1, H = 2, W = 2;
        const int size = N * C * H * W;
        float h_input[]  = {1.0f, 2.0f, 3.0f, 4.0f};
        float h_output[1];  // Output will be a single value after 2x2 pooling

        float *d_input, *d_output;
        CUDA_CHECK(hipMalloc(&d_input,  size * sizeof(float)));
        CUDA_CHECK(hipMalloc(&d_output, sizeof(float)));

        CUDA_CHECK(hipMemcpy(d_input, h_input, sizeof(h_input), hipMemcpyHostToDevice));

        hipdnnTensorDescriptor_t inputDesc, outputDesc;
        CUDNN_CHECK(hipdnnCreateTensorDescriptor(&inputDesc));
        CUDNN_CHECK(hipdnnCreateTensorDescriptor(&outputDesc));

        // Set tensor descriptors
        CUDNN_CHECK(hipdnnSetTensor4dDescriptor(inputDesc, HIPDNN_TENSOR_NCHW, HIPDNN_DATA_FLOAT, N, C, H, W));
        CUDNN_CHECK(hipdnnSetTensor4dDescriptor(outputDesc, HIPDNN_TENSOR_NCHW, HIPDNN_DATA_FLOAT, N, C, 1, 1));

        // Create and set pooling descriptor
        hipdnnPoolingDescriptor_t poolingDesc;
        CUDNN_CHECK(hipdnnCreatePoolingDescriptor(&poolingDesc));

        CUDNN_CHECK(hipdnnSetPooling2dDescriptor(poolingDesc,
                                                HIPDNN_POOLING_MAX,
                                                HIPDNN_PROPAGATE_NAN,
                                                2, 2,   // window height, width
                                                0, 0,   // padding height, width
                                                2, 2)); // stride height, width

        float alpha = 1.0f, beta = 0.0f;
        CUDNN_CHECK(hipdnnPoolingForward(handle,
                                        poolingDesc,
                                        &alpha,
                                        inputDesc, d_input,
                                        &beta,
                                        outputDesc, d_output));

        CUDA_CHECK(hipMemcpy(h_output, d_output, sizeof(float), hipMemcpyDeviceToHost));

        // Expected output is max(1, 2, 3, 4) = 4.0f
        EXPECT_FLOAT_EQ(h_output[0], 4.0f);

        // Cleanup
        CUDNN_CHECK(hipdnnDestroyPoolingDescriptor(poolingDesc));
        CUDNN_CHECK(hipdnnDestroyTensorDescriptor(inputDesc));
        CUDNN_CHECK(hipdnnDestroyTensorDescriptor(outputDesc));
        CUDA_CHECK(hipFree(d_input));
        CUDA_CHECK(hipFree(d_output));
        CUDNN_CHECK(hipdnnDestroy(handle));
    });
}

TEST_F(CuDNNTestWithCatch, ConvolutionForward) {
    RunWithExceptionHandling([](){
        hipdnnHandle_t handle;
        CUDNN_CHECK(hipdnnCreate(&handle));

        const int N = 1, C = 1, H = 2, W = 2;
        const int size = N * C * H * W;

        float h_input[]  = {1.0f, 2.0f, 3.0f, 4.0f};
        float h_filter[] = {10.0f};  // 1x1 filter
        float h_output[4];

        float *d_input, *d_filter, *d_output;
        CUDA_CHECK(hipMalloc(&d_input,  size * sizeof(float)));
        CUDA_CHECK(hipMalloc(&d_filter, sizeof(float)));
        CUDA_CHECK(hipMalloc(&d_output, size * sizeof(float)));

        CUDA_CHECK(hipMemcpy(d_input, h_input, sizeof(h_input), hipMemcpyHostToDevice));
        CUDA_CHECK(hipMemcpy(d_filter, h_filter, sizeof(h_filter), hipMemcpyHostToDevice));

        // Create descriptors
        hipdnnTensorDescriptor_t inputDesc, outputDesc;
        hipdnnFilterDescriptor_t filterDesc;
        hipdnnConvolutionDescriptor_t convDesc;

        CUDNN_CHECK(hipdnnCreateTensorDescriptor(&inputDesc));
        CUDNN_CHECK(hipdnnCreateTensorDescriptor(&outputDesc));
        CUDNN_CHECK(hipdnnCreateFilterDescriptor(&filterDesc));
        CUDNN_CHECK(hipdnnCreateConvolutionDescriptor(&convDesc));

        CUDNN_CHECK(hipdnnSetTensor4dDescriptor(inputDesc, HIPDNN_TENSOR_NCHW, HIPDNN_DATA_FLOAT, N, C, H, W));
        CUDNN_CHECK(hipdnnSetTensor4dDescriptor(outputDesc, HIPDNN_TENSOR_NCHW, HIPDNN_DATA_FLOAT, N, C, H, W));
        CUDNN_CHECK(hipdnnSetFilter4dDescriptor(filterDesc, HIPDNN_DATA_FLOAT, HIPDNN_TENSOR_NCHW, 1, 1, 1, 1));

        CUDNN_CHECK(hipdnnSetConvolution2dDescriptor(convDesc, 0, 0, 1, 1, 1, 1, HIPDNN_CROSS_CORRELATION, HIPDNN_DATA_FLOAT));

        // Choose algo
        hipdnnConvolutionFwdAlgo_t algo = HIPDNN_CONVOLUTION_FWD_ALGO_IMPLICIT_PRECOMP_GEMM;

        // Workspace
        size_t workspaceBytes = 0;
        CUDNN_CHECK(hipdnnGetConvolutionForwardWorkspaceSize(handle, inputDesc, filterDesc, convDesc, outputDesc, algo, &workspaceBytes));

        void* d_workspace = nullptr;
        if (workspaceBytes > 0)
            CUDA_CHECK(hipMalloc(&d_workspace, workspaceBytes));

        float alpha = 1.0f, beta = 0.0f;
        CUDNN_CHECK(hipdnnConvolutionForward(handle,
                                            &alpha,
                                            inputDesc, d_input,
                                            filterDesc, d_filter,
                                            convDesc, algo,
                                            d_workspace, workspaceBytes,
                                            &beta,
                                            outputDesc, d_output));

        CUDA_CHECK(hipMemcpy(h_output, d_output, sizeof(h_output), hipMemcpyDeviceToHost));

        // Verify results
        EXPECT_FLOAT_EQ(h_output[0], 10.0f);
        EXPECT_FLOAT_EQ(h_output[1], 20.0f);
        EXPECT_FLOAT_EQ(h_output[2], 30.0f);
        EXPECT_FLOAT_EQ(h_output[3], 40.0f);

        // Cleanup
        if (workspaceBytes > 0)
            CUDA_CHECK(hipFree(d_workspace));

        CUDNN_CHECK(hipdnnDestroyTensorDescriptor(inputDesc));
        CUDNN_CHECK(hipdnnDestroyTensorDescriptor(outputDesc));
        CUDNN_CHECK(hipdnnDestroyFilterDescriptor(filterDesc));
        CUDNN_CHECK(hipdnnDestroyConvolutionDescriptor(convDesc));
        CUDA_CHECK(hipFree(d_input));
        CUDA_CHECK(hipFree(d_filter));
        CUDA_CHECK(hipFree(d_output));
        CUDNN_CHECK(hipdnnDestroy(handle));
    });
}

TEST_F(CuDNNTestWithCatch, FilterDescriptorCreateSetGet) {
    RunWithExceptionHandling([](){
        hipdnnHandle_t handle;
        CUDNN_CHECK(hipdnnCreate(&handle));

        hipdnnFilterDescriptor_t filterDesc;
        CUDNN_CHECK(hipdnnCreateFilterDescriptor(&filterDesc));

        // Set descriptor: format NCHW, 1 output, 1 input, 3x3 kernel
        const int k = 1, c = 1, h = 3, w = 3;
        CUDNN_CHECK(hipdnnSetFilter4dDescriptor(filterDesc, HIPDNN_DATA_FLOAT, HIPDNN_TENSOR_NCHW, k, c, h, w));

        // Retrieve and check descriptor values
        hipdnnDataType_t dataType;
        hipdnnTensorFormat_t format;
        int k_ret, c_ret, h_ret, w_ret;
        CUDNN_CHECK(hipdnnGetFilter4dDescriptor(filterDesc, &dataType, &format, &k_ret, &c_ret, &h_ret, &w_ret));

        EXPECT_EQ(dataType, HIPDNN_DATA_FLOAT);
        EXPECT_EQ(format,  HIPDNN_TENSOR_NCHW);
        EXPECT_EQ(k_ret, k);
        EXPECT_EQ(c_ret, c);
        EXPECT_EQ(h_ret, h);
        EXPECT_EQ(w_ret, w);

        // Cleanup
        CUDNN_CHECK(hipdnnDestroyFilterDescriptor(filterDesc));
        CUDNN_CHECK(hipdnnDestroy(handle));
    });
}

TEST_F(CuDNNTestWithCatch, LRNForward) {
    RunWithExceptionHandling([](){
    hipdnnHandle_t handle;
    CUDNN_CHECK(hipdnnCreate(&handle));

    // Tensor dims: NCHW = 1x1x1x5
    const int N = 1, C = 5, H = 1, W = 1;
    const int size = N * C * H * W;

    float h_input[]  = {1.0f, 2.0f, 3.0f, 4.0f, 5.0f};
    float h_output[size];

    float *d_input, *d_output;
    CUDA_CHECK(hipMalloc(&d_input,  size * sizeof(float)));
    CUDA_CHECK(hipMalloc(&d_output, size * sizeof(float)));

    CUDA_CHECK(hipMemcpy(d_input, h_input, sizeof(h_input), hipMemcpyHostToDevice));

    // Create tensor descriptors
    hipdnnTensorDescriptor_t tensorDesc;
    CUDNN_CHECK(hipdnnCreateTensorDescriptor(&tensorDesc));
    CUDNN_CHECK(hipdnnSetTensor4dDescriptor(tensorDesc,
                                           HIPDNN_TENSOR_NCHW,
                                           HIPDNN_DATA_FLOAT,
                                           N, C, H, W));

    // Create LRN descriptor
    hipdnnLRNDescriptor_t lrnDesc;
    CUDNN_CHECK(hipdnnCreateLRNDescriptor(&lrnDesc));

    // Set LRN parameters: local_size, alpha, beta, k
    const unsigned localSize = 3;
    const double alpha = 1e-4;
    const double beta  = 0.75;
    const double k     = 2.0;

    CUDNN_CHECK(hipdnnSetLRNDescriptor(lrnDesc, localSize, alpha, beta, k));

    float alpha1 = 1.0f, beta1 = 0.0f;
    CUDNN_CHECK(hipdnnLRNCrossChannelForward(handle,
                                            lrnDesc,
                                            HIPDNN_LRN_CROSS_CHANNEL,
                                            &alpha1,
                                            tensorDesc, d_input,
                                            &beta1,
                                            tensorDesc, d_output));

    CUDA_CHECK(hipMemcpy(h_output, d_output, sizeof(h_output), hipMemcpyDeviceToHost));

    // Print results (since exact analytical value is tedious, we can sanity check)
    for (int i = 0; i < size; ++i) {
        printf("LRN output[%d] = %f\n", i, h_output[i]);
    }

    // Basic sanity check: output should be less than or equal to input since normalization happens
    for (int i = 0; i < size; ++i) {
        EXPECT_LE(h_output[i], h_input[i]);
    }

    // Cleanup
    CUDNN_CHECK(hipdnnDestroyLRNDescriptor(lrnDesc));
    CUDNN_CHECK(hipdnnDestroyTensorDescriptor(tensorDesc));
    CUDA_CHECK(hipFree(d_input));
    CUDA_CHECK(hipFree(d_output));
    CUDNN_CHECK(hipdnnDestroy(handle));
    });
}

/*
TEST(cuDNN, CreateDestroy) {
    hipdnnHandle_t handle;
    CUDNN_CHECK(hipdnnCreate(&handle));
    CUDNN_CHECK(hipdnnDestroy(handle));
}

TEST(cuDNN, SetStreamDestroy) {
    hipdnnHandle_t handle;
    hipStream_t stream;
    CUDNN_CHECK(hipdnnCreate(&handle));
    CUDA_CHECK(hipStreamCreate(&stream));
    CUDNN_CHECK(hipdnnSetStream(handle, stream));
    CUDA_CHECK(hipStreamDestroy(stream));
    CUDNN_CHECK(hipdnnDestroy(handle));
}

TEST(cuDNN, AddTensor) {
    hipdnnHandle_t handle;
    CUDNN_CHECK(hipdnnCreate(&handle));

    const int N = 1, C = 1, H = 2, W = 2;
    const int size = N * C * H * W;
    float h_A[] = {1, 2, 3, 4};
    float h_B[] = {10, 20, 30, 40};

    float *d_A, *d_B;
    CUDA_CHECK(hipMalloc(&d_A, size * sizeof(float)));
    CUDA_CHECK(hipMalloc(&d_B, size * sizeof(float)));

    CUDA_CHECK(hipMemcpy(d_A, h_A, sizeof(h_A), hipMemcpyHostToDevice));
    CUDA_CHECK(hipMemcpy(d_B, h_B, sizeof(h_B), hipMemcpyHostToDevice));

    hipdnnTensorDescriptor_t desc;
    CUDNN_CHECK(hipdnnCreateTensorDescriptor(&desc));
    CUDNN_CHECK(hipdnnSetTensor4dDescriptor(desc, 
                                           HIPDNN_TENSOR_NCHW, 
                                           HIPDNN_DATA_FLOAT, 
                                           N, C, H, W));

    float alpha = 1.0f;
    float beta  = 1.0f;

    // B = alpha * A + beta * B
    CUDNN_CHECK(hipdnnAddTensor(handle,
                               &alpha,
                               desc, d_A,
                               &beta,
                               desc, d_B));

    float h_result[size];
    CUDA_CHECK(hipMemcpy(h_result, d_B, sizeof(h_result), hipMemcpyDeviceToHost));

    // Expected result: B[i] = A[i] + B[i]
    EXPECT_FLOAT_EQ(h_result[0], 11.0f);
    EXPECT_FLOAT_EQ(h_result[1], 22.0f);
    EXPECT_FLOAT_EQ(h_result[2], 33.0f);
    EXPECT_FLOAT_EQ(h_result[3], 44.0f);

    CUDNN_CHECK(hipdnnDestroyTensorDescriptor(desc));
    CUDA_CHECK(hipFree(d_A));
    CUDA_CHECK(hipFree(d_B));
    CUDNN_CHECK(hipdnnDestroy(handle));
}

TEST(cuDNN, PoolingForward) {
    hipdnnHandle_t handle;
    CUDNN_CHECK(hipdnnCreate(&handle));

    const int N = 1, C = 1, H = 2, W = 2;
    const int size = N * C * H * W;
    float h_input[]  = {1.0f, 2.0f, 3.0f, 4.0f};
    float h_output[1];  // Output will be a single value after 2x2 pooling

    float *d_input, *d_output;
    CUDA_CHECK(hipMalloc(&d_input,  size * sizeof(float)));
    CUDA_CHECK(hipMalloc(&d_output, sizeof(float)));

    CUDA_CHECK(hipMemcpy(d_input, h_input, sizeof(h_input), hipMemcpyHostToDevice));

    hipdnnTensorDescriptor_t inputDesc, outputDesc;
    CUDNN_CHECK(hipdnnCreateTensorDescriptor(&inputDesc));
    CUDNN_CHECK(hipdnnCreateTensorDescriptor(&outputDesc));

    // Input: NCHW format
    CUDNN_CHECK(hipdnnSetTensor4dDescriptor(inputDesc, HIPDNN_TENSOR_NCHW, HIPDNN_DATA_FLOAT, N, C, H, W));
    // Output after 2x2 pooling with stride 2 will be 1x1
    CUDNN_CHECK(hipdnnSetTensor4dDescriptor(outputDesc, HIPDNN_TENSOR_NCHW, HIPDNN_DATA_FLOAT, N, C, 1, 1));

    // Create pooling descriptor
    hipdnnPoolingDescriptor_t poolingDesc;
    CUDNN_CHECK(hipdnnCreatePoolingDescriptor(&poolingDesc));

    CUDNN_CHECK(hipdnnSetPooling2dDescriptor(poolingDesc,
                                            HIPDNN_POOLING_MAX,
                                            HIPDNN_PROPAGATE_NAN,
                                            2, 2,   // window height, width
                                            0, 0,   // padding height, width
                                            2, 2)); // stride height, width

    float alpha = 1.0f, beta = 0.0f;
    CUDNN_CHECK(hipdnnPoolingForward(handle,
                                    poolingDesc,
                                    &alpha,
                                    inputDesc, d_input,
                                    &beta,
                                    outputDesc, d_output));

    CUDA_CHECK(hipMemcpy(h_output, d_output, sizeof(float), hipMemcpyDeviceToHost));

    // Expected output is max(1, 2, 3, 4) = 4.0f
    EXPECT_FLOAT_EQ(h_output[0], 4.0f);

    // Cleanup
    CUDNN_CHECK(hipdnnDestroyPoolingDescriptor(poolingDesc));
    CUDNN_CHECK(hipdnnDestroyTensorDescriptor(inputDesc));
    CUDNN_CHECK(hipdnnDestroyTensorDescriptor(outputDesc));
    CUDA_CHECK(hipFree(d_input));
    CUDA_CHECK(hipFree(d_output));
    CUDNN_CHECK(hipdnnDestroy(handle));
}

TEST(cuDNN, ConvolutionForward) {
    hipdnnHandle_t handle;
    CUDNN_CHECK(hipdnnCreate(&handle));

    const int N = 1, C = 1, H = 2, W = 2;
    const int size = N * C * H * W;

    float h_input[]  = {1.0f, 2.0f, 3.0f, 4.0f};
    float h_filter[] = {10.0f};  // 1x1 filter with value 10
    float h_output[4];

    float *d_input, *d_filter, *d_output;
    CUDA_CHECK(hipMalloc(&d_input,  size * sizeof(float)));
    CUDA_CHECK(hipMalloc(&d_filter, sizeof(float)));
    CUDA_CHECK(hipMalloc(&d_output, size * sizeof(float)));

    CUDA_CHECK(hipMemcpy(d_input, h_input, sizeof(h_input), hipMemcpyHostToDevice));
    CUDA_CHECK(hipMemcpy(d_filter, h_filter, sizeof(h_filter), hipMemcpyHostToDevice));

    // Create descriptors
    hipdnnTensorDescriptor_t inputDesc, outputDesc;
    hipdnnFilterDescriptor_t filterDesc;
    hipdnnConvolutionDescriptor_t convDesc;

    CUDNN_CHECK(hipdnnCreateTensorDescriptor(&inputDesc));
    CUDNN_CHECK(hipdnnCreateTensorDescriptor(&outputDesc));
    CUDNN_CHECK(hipdnnCreateFilterDescriptor(&filterDesc));
    CUDNN_CHECK(hipdnnCreateConvolutionDescriptor(&convDesc));

    CUDNN_CHECK(hipdnnSetTensor4dDescriptor(inputDesc, HIPDNN_TENSOR_NCHW, HIPDNN_DATA_FLOAT, N, C, H, W));
    CUDNN_CHECK(hipdnnSetTensor4dDescriptor(outputDesc, HIPDNN_TENSOR_NCHW, HIPDNN_DATA_FLOAT, N, C, H, W));

    CUDNN_CHECK(hipdnnSetFilter4dDescriptor(filterDesc, HIPDNN_DATA_FLOAT, HIPDNN_TENSOR_NCHW, 1, 1, 1, 1));

    // Set conv descriptor: zero padding, stride 1, dilation 1
    CUDNN_CHECK(hipdnnSetConvolution2dDescriptor(convDesc,
                                                0, 0,   // pad_h, pad_w
                                                1, 1,   // stride_h, stride_w
                                                1, 1,   // dilation_h, dilation_w
                                                HIPDNN_CROSS_CORRELATION, 
                                                HIPDNN_DATA_FLOAT));

    // Select algorithm (fastest or deterministic)
    hipdnnConvolutionFwdAlgo_t algo = HIPDNN_CONVOLUTION_FWD_ALGO_IMPLICIT_PRECOMP_GEMM;

    // Get workspace size
    size_t workspaceBytes = 0;
    CUDNN_CHECK(hipdnnGetConvolutionForwardWorkspaceSize(handle,
                                                        inputDesc, filterDesc, convDesc, outputDesc,
                                                        algo, &workspaceBytes));

    void* d_workspace = nullptr;
    if (workspaceBytes > 0)
        CUDA_CHECK(hipMalloc(&d_workspace, workspaceBytes));

    // Launch convolution
    float alpha = 1.0f, beta = 0.0f;
    CUDNN_CHECK(hipdnnConvolutionForward(handle,
                                        &alpha,
                                        inputDesc, d_input,
                                        filterDesc, d_filter,
                                        convDesc, algo,
                                        d_workspace, workspaceBytes,
                                        &beta,
                                        outputDesc, d_output));

    // Copy result back
    CUDA_CHECK(hipMemcpy(h_output, d_output, sizeof(h_output), hipMemcpyDeviceToHost));

    // Expected: each input value multiplied by 10
    EXPECT_FLOAT_EQ(h_output[0], 10.0f);
    EXPECT_FLOAT_EQ(h_output[1], 20.0f);
    EXPECT_FLOAT_EQ(h_output[2], 30.0f);
    EXPECT_FLOAT_EQ(h_output[3], 40.0f);

    // Cleanup
    if (workspaceBytes > 0)
        CUDA_CHECK(hipFree(d_workspace));

    CUDNN_CHECK(hipdnnDestroyTensorDescriptor(inputDesc));
    CUDNN_CHECK(hipdnnDestroyTensorDescriptor(outputDesc));
    CUDNN_CHECK(hipdnnDestroyFilterDescriptor(filterDesc));
    CUDNN_CHECK(hipdnnDestroyConvolutionDescriptor(convDesc));
    CUDA_CHECK(hipFree(d_input));
    CUDA_CHECK(hipFree(d_filter));
    CUDA_CHECK(hipFree(d_output));
    CUDNN_CHECK(hipdnnDestroy(handle));
}

TEST(cuDNN, FilterDescriptorCreateSetGet) {
    hipdnnHandle_t handle;
    CUDNN_CHECK(hipdnnCreate(&handle));

    hipdnnFilterDescriptor_t filterDesc;
    CUDNN_CHECK(hipdnnCreateFilterDescriptor(&filterDesc));

    // Set descriptor: format NCHW, 1 output channel, 1 input channel, 3x3 kernel
    const int k = 1;  // output channels
    const int c = 1;  // input channels
    const int h = 3;
    const int w = 3;
    CUDNN_CHECK(hipdnnSetFilter4dDescriptor(filterDesc,
                                           HIPDNN_DATA_FLOAT,
                                           HIPDNN_TENSOR_NCHW,
                                           k, c, h, w));

    // Retrieve and check descriptor values
    hipdnnDataType_t dataType;
    hipdnnTensorFormat_t format;
    int k_ret, c_ret, h_ret, w_ret;
    CUDNN_CHECK(hipdnnGetFilter4dDescriptor(filterDesc,
                                           &dataType,
                                           &format,
                                           &k_ret, &c_ret, &h_ret, &w_ret));

    EXPECT_EQ(dataType, HIPDNN_DATA_FLOAT);
    EXPECT_EQ(format,  HIPDNN_TENSOR_NCHW);
    EXPECT_EQ(k_ret, k);
    EXPECT_EQ(c_ret, c);
    EXPECT_EQ(h_ret, h);
    EXPECT_EQ(w_ret, w);

    // Clean up
    CUDNN_CHECK(hipdnnDestroyFilterDescriptor(filterDesc));
    CUDNN_CHECK(hipdnnDestroy(handle));
}

TEST(cuDNN, LRNForward) {
    hipdnnHandle_t handle;
    CUDNN_CHECK(hipdnnCreate(&handle));

    // Tensor dims: NCHW = 1x1x1x5
    const int N = 1, C = 5, H = 1, W = 1;
    const int size = N * C * H * W;

    float h_input[]  = {1.0f, 2.0f, 3.0f, 4.0f, 5.0f};
    float h_output[size];

    float *d_input, *d_output;
    CUDA_CHECK(hipMalloc(&d_input,  size * sizeof(float)));
    CUDA_CHECK(hipMalloc(&d_output, size * sizeof(float)));

    CUDA_CHECK(hipMemcpy(d_input, h_input, sizeof(h_input), hipMemcpyHostToDevice));

    // Create tensor descriptors
    hipdnnTensorDescriptor_t tensorDesc;
    CUDNN_CHECK(hipdnnCreateTensorDescriptor(&tensorDesc));
    CUDNN_CHECK(hipdnnSetTensor4dDescriptor(tensorDesc,
                                           HIPDNN_TENSOR_NCHW,
                                           HIPDNN_DATA_FLOAT,
                                           N, C, H, W));

    // Create LRN descriptor
    hipdnnLRNDescriptor_t lrnDesc;
    CUDNN_CHECK(hipdnnCreateLRNDescriptor(&lrnDesc));

    // Set LRN parameters: local_size, alpha, beta, k
    const unsigned localSize = 3;
    const double alpha = 1e-4;
    const double beta  = 0.75;
    const double k     = 2.0;

    CUDNN_CHECK(hipdnnSetLRNDescriptor(lrnDesc, localSize, alpha, beta, k));

    float alpha1 = 1.0f, beta1 = 0.0f;
    CUDNN_CHECK(hipdnnLRNCrossChannelForward(handle,
                                            lrnDesc,
                                            HIPDNN_LRN_CROSS_CHANNEL,
                                            &alpha1,
                                            tensorDesc, d_input,
                                            &beta1,
                                            tensorDesc, d_output));

    CUDA_CHECK(hipMemcpy(h_output, d_output, sizeof(h_output), hipMemcpyDeviceToHost));

    // Print results (since exact analytical value is tedious, we can sanity check)
    for (int i = 0; i < size; ++i) {
        printf("LRN output[%d] = %f\n", i, h_output[i]);
    }

    // Basic sanity check: output should be less than or equal to input since normalization happens
    for (int i = 0; i < size; ++i) {
        EXPECT_LE(h_output[i], h_input[i]);
    }

    // Cleanup
    CUDNN_CHECK(hipdnnDestroyLRNDescriptor(lrnDesc));
    CUDNN_CHECK(hipdnnDestroyTensorDescriptor(tensorDesc));
    CUDA_CHECK(hipFree(d_input));
    CUDA_CHECK(hipFree(d_output));
    CUDNN_CHECK(hipdnnDestroy(handle));
}

*/
