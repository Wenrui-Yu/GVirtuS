#include <iostream>
#include <stdexcept>
#include <hipDNN.h>
#include <hip/hip_runtime.h>
#include "cudnn_utils.h" // 确保这里的 checkCUDNN / checkCUDA 已定义为抛出 std::runtime_error

int main() {
    try {
        hipdnnHandle_t cudnn;
        checkCUDNN(hipdnnCreate(&cudnn));

        hipdnnLRNDescriptor_t lrnDesc;
        checkCUDNN(hipdnnCreateLRNDescriptor(&lrnDesc));
        checkCUDNN(hipdnnSetLRNDescriptor(lrnDesc, 5, 1e-4, 0.75, 2.0));

        hipdnnTensorDescriptor_t tensorDesc;
        checkCUDNN(hipdnnCreateTensorDescriptor(&tensorDesc));
        checkCUDNN(hipdnnSetTensor4dDescriptor(tensorDesc, HIPDNN_TENSOR_NCHW, HIPDNN_DATA_FLOAT, 1, 1, 3, 3));

        float h_input[9] = {
            1.0f, 2.0f, 3.0f,
            4.0f, 5.0f, 6.0f,
            7.0f, 8.0f, 9.0f
        };
        float h_output[9] = {0};

        float *d_input = nullptr, *d_output = nullptr;
        checkCUDA(hipMalloc(&d_input, sizeof(h_input)));
        checkCUDA(hipMalloc(&d_output, sizeof(h_output)));
        checkCUDA(hipMemcpy(d_input, h_input, sizeof(h_input), hipMemcpyHostToDevice));

        float alpha = 1.0f, beta = 0.0f;

        checkCUDNN(hipdnnLRNCrossChannelForward(
            cudnn,
            lrnDesc,
            HIPDNN_LRN_CROSS_CHANNEL,
            &alpha,
            tensorDesc,
            d_input,
            &beta,
            tensorDesc,
            d_output
        ));

        checkCUDA(hipMemcpy(h_output, d_output, sizeof(h_output), hipMemcpyDeviceToHost));

        std::cout << "LRN output:" << std::endl;
        for (int i = 0; i < 9; ++i) {
            std::cout << h_output[i] << " ";
            if ((i + 1) % 3 == 0) std::cout << std::endl;
        }

        // 清理资源
        checkCUDA(hipFree(d_input));
        checkCUDA(hipFree(d_output));
        checkCUDNN(hipdnnDestroyTensorDescriptor(tensorDesc));
        checkCUDNN(hipdnnDestroyLRNDescriptor(lrnDesc));
        checkCUDNN(hipdnnDestroy(cudnn));

        return 0;
    }
    catch (const std::exception& e) {
        std::cerr << "Caught std::exception: " << e.what() << std::endl;
        return EXIT_FAILURE;
    }
    catch (...) {
        std::cerr << "Caught unknown exception." << std::endl;
        return EXIT_FAILURE;
    }
}
