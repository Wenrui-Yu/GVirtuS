#include <iostream>
#include <hip/hip_runtime.h>
#include <cassert>

int main() {
    int deviceCount = 0;
    hipError_t err = hipGetDeviceCount(&deviceCount);

    if (err != hipSuccess) {
        std::cerr << "hipGetDeviceCount failed: "
                  << hipGetErrorString(err) << std::endl;
        return 1;
    }

    std::cout << "Number of CUDA devices: " << deviceCount << std::endl;
    assert(deviceCount >= 0);  // Sanity check

    return 0;
}
