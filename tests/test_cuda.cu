#include "hip/hip_runtime.h"
#include <gtest/gtest.h>
#include <hip/hip_runtime.h>

#define CUDA_CHECK(stmt) ASSERT_EQ((stmt), hipSuccess)

// __global__ void incrementKernel(int* data) {
//     int idx = threadIdx.x;
//     data[idx] += 1;
// }


TEST(CUDA_Runtime, EventTiming) {
    hipEvent_t start, stop;
    CUDA_CHECK(hipEventCreate(&start));
    CUDA_CHECK(hipEventCreate(&stop));
    CUDA_CHECK(hipEventRecord(start));
    CUDA_CHECK(hipEventRecord(stop));
    CUDA_CHECK(hipEventSynchronize(stop));
    float ms = 0.0f;
    CUDA_CHECK(hipEventElapsedTime(&ms, start, stop));
    EXPECT_GE(ms, 0.0f);
    CUDA_CHECK(hipEventDestroy(start));
    CUDA_CHECK(hipEventDestroy(stop));
}

TEST(CUDA_Runtime, MallocMemcpyMemsetFree) {
    const size_t n = 100;
    int *d_ptr;
    CUDA_CHECK(hipMalloc(&d_ptr, n * sizeof(int)));
    CUDA_CHECK(hipMemset(d_ptr, 0, n * sizeof(int)));
    int h_data[n] = {1};
    CUDA_CHECK(hipMemcpy(d_ptr, h_data, sizeof(h_data), hipMemcpyHostToDevice));
    CUDA_CHECK(hipFree(d_ptr));
}

TEST(CUDA_Runtime, MallocHostFreeHost) {
    int* h_ptr = nullptr;
    CUDA_CHECK(hipHostMalloc(&h_ptr, 100 * sizeof(int)));
    ASSERT_NE(h_ptr, nullptr);
    CUDA_CHECK(hipHostFree(h_ptr));
}

TEST(CUDA_Runtime, MemcpyAsyncWithStream) {
    const int n = 10;
    int h_src[n] = {1,2,3,4,5,6,7,8,9,10};
    int h_dst[n] = {0};
    int *d_ptr;
    hipStream_t stream;
    CUDA_CHECK(hipMalloc(&d_ptr, sizeof(h_src)));
    CUDA_CHECK(hipStreamCreate(&stream));
    CUDA_CHECK(hipMemcpyAsync(d_ptr, h_src, sizeof(h_src), hipMemcpyHostToDevice, stream));
    CUDA_CHECK(hipMemcpyAsync(h_dst, d_ptr, sizeof(h_dst), hipMemcpyDeviceToHost, stream));
    CUDA_CHECK(hipStreamSynchronize(stream));
    for (int i = 0; i < n; ++i) EXPECT_EQ(h_src[i], h_dst[i]);
    CUDA_CHECK(hipStreamDestroy(stream));
    CUDA_CHECK(hipFree(d_ptr));
}

TEST(CUDA_Runtime, StreamCreateWithFlags) {
    hipStream_t stream;
    CUDA_CHECK(hipStreamCreateWithFlags(&stream, hipStreamNonBlocking));
    CUDA_CHECK(hipStreamDestroy(stream));
}

TEST(CUDA_Runtime, DeviceInfo) {
    int count = 0;
    CUDA_CHECK(hipGetDeviceCount(&count));
    ASSERT_GT(count, 0);
    CUDA_CHECK(hipSetDevice(0));
    int current = -1;
    CUDA_CHECK(hipGetDevice(&current));
    EXPECT_EQ(current, 0);
    hipDeviceProp_t prop;
    CUDA_CHECK(hipGetDeviceProperties(&prop, 0));
    ASSERT_GT(prop.totalGlobalMem, 0);
}

TEST(CUDA_Runtime, MemGetInfo) {
    size_t freeMem = 0, totalMem = 0;
    CUDA_CHECK(hipMemGetInfo(&freeMem, &totalMem));
    ASSERT_GT(freeMem, 0);
    ASSERT_GT(totalMem, 0);
}

TEST(CUDA_Runtime, ErrorQuerying) {
    hipError_t peek = hipPeekAtLastError();
    EXPECT_EQ(peek, hipSuccess);
    hipError_t last = hipGetLastError();
    EXPECT_EQ(last, hipSuccess);
    const char* errStr = hipGetErrorString(last);
    ASSERT_NE(errStr, nullptr);
}

__global__ void addOneKernel(int* data) {
    int idx = threadIdx.x;
    data[idx] += 1;
}

TEST(CUDA_Runtime, LaunchKernel) {
    const int N = 4;
    int h_data[N] = {0, 1, 2, 3};
    int* d_data = nullptr;

    CUDA_CHECK(hipMalloc(&d_data, sizeof(h_data)));
    CUDA_CHECK(hipMemcpy(d_data, h_data, sizeof(h_data), hipMemcpyHostToDevice));

    // Launch the kernel using the standard CUDA kernel call syntax
    addOneKernel<<<1, N>>>(d_data);
    CUDA_CHECK(hipDeviceSynchronize());

    CUDA_CHECK(hipMemcpy(h_data, d_data, sizeof(h_data), hipMemcpyDeviceToHost));
    EXPECT_EQ(h_data[0], 1);
    EXPECT_EQ(h_data[3], 4);

    CUDA_CHECK(hipFree(d_data));
}



TEST(CUDA_Runtime, EventCreateDestroy) {
    hipEvent_t evt;
    CUDA_CHECK(hipEventCreate(&evt));
    CUDA_CHECK(hipEventDestroy(evt));
}

TEST(CUDA_Runtime, EventRecordAndSync) {
    hipEvent_t evt;
    CUDA_CHECK(hipEventCreate(&evt));
    CUDA_CHECK(hipEventRecord(evt));
    CUDA_CHECK(hipEventSynchronize(evt));
    CUDA_CHECK(hipEventDestroy(evt));
}

TEST(CUDA_Runtime, GetSetDevice) {
    int devCount = 0;
    CUDA_CHECK(hipGetDeviceCount(&devCount));
    ASSERT_GT(devCount, 0);
    CUDA_CHECK(hipSetDevice(0));
    int current;
    CUDA_CHECK(hipGetDevice(&current));
    EXPECT_EQ(current, 0);
}

TEST(CUDA_Runtime, GetDeviceProperties) {
    hipDeviceProp_t prop;
    CUDA_CHECK(hipGetDeviceProperties(&prop, 0));
    EXPECT_GT(prop.totalGlobalMem, 0);
}

TEST(CUDA_Runtime, GetErrorInfo) {
    // Force an error by launching an invalid kernel configuration
    hipError_t err = hipMemcpy(nullptr, nullptr, 10, hipMemcpyHostToDevice); // Invalid
    EXPECT_NE(err, hipSuccess);

    // Check GetLastError and GetErrorString
    hipError_t last = hipGetLastError();
    EXPECT_EQ(last, err);

    const char* errStr = hipGetErrorString(last);
    ASSERT_NE(errStr, nullptr);
    std::cout << "Captured CUDA error: " << errStr << std::endl;

    // Clear error
    EXPECT_EQ(hipGetLastError(), hipSuccess);
}
