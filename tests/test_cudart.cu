#include "hip/hip_runtime.h"
#include <gtest/gtest.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h> /* cuuint64_t */

#define CUDA_CHECK(err) ASSERT_EQ((err), hipSuccess)

__device__ int intDeviceVariable = 0;

//Unit tests for only those functions which are missing with openpose integration!

//hipFree
TEST(cudaRT, MallocAndFree) {
    int* device_ptr = nullptr;

    ASSERT_EQ(hipMalloc((void**)&device_ptr, 256 * sizeof(int)), hipSuccess);
    ASSERT_NE(device_ptr, nullptr);

    ASSERT_EQ(hipFree(device_ptr), hipSuccess);
}

//StreamCreateWithFlagsAndDestroy
TEST(cudaRT, StreamCreateWithFlagsAndDestroy) {
    hipStream_t stream;
    ASSERT_EQ(hipStreamCreateWithFlags(&stream, hipStreamNonBlocking), hipSuccess);
    ASSERT_NE(stream, nullptr);
    ASSERT_EQ(hipStreamDestroy(stream), hipSuccess);
}


// RegisterFatBinaryIndirectCheck
// __global__ void dummyKernel(int* data) {
//     data[0] = 123;
// }

// TEST(cudaRT, RegisterFatBinaryIndirectCheck) {
//     int* d_ptr;
//     ASSERT_EQ(hipMalloc(&d_ptr, sizeof(int)), hipSuccess);
//     dummyKernel<<<1, 1>>>(d_ptr);
//     ASSERT_EQ(hipDeviceSynchronize(), hipSuccess);
//     ASSERT_EQ(hipFree(d_ptr), hipSuccess);
// }


//hipEventElapsedTime
TEST(cudaRT, EventElapsedTime) {
    hipEvent_t start, stop;
    float elapsed_ms = 0.0f;

    ASSERT_EQ(hipEventCreate(&start), hipSuccess);
    ASSERT_EQ(hipEventCreate(&stop), hipSuccess);

    ASSERT_EQ(hipEventRecord(start), hipSuccess);
    ASSERT_EQ(hipEventRecord(stop), hipSuccess);
    ASSERT_EQ(hipEventSynchronize(stop), hipSuccess);

    ASSERT_EQ(hipEventElapsedTime(&elapsed_ms, start, stop), hipSuccess);
    ASSERT_GT(elapsed_ms, 0.0f);

    ASSERT_EQ(hipEventDestroy(start), hipSuccess);
    ASSERT_EQ(hipEventDestroy(stop), hipSuccess);
}

//__cudaRegisterFunction
// __global__ void dummyKernel(int* out) {
//     *out = 42;
// }

// TEST(cudaRT, RegisterFunctionIndirectCheck) {
//     int* d_ptr;
//     int h_val = 0;

//     ASSERT_EQ(hipMalloc(&d_ptr, sizeof(int)), hipSuccess);
//     dummyKernel<<<1, 1>>>(d_ptr);
//     ASSERT_EQ(hipDeviceSynchronize(), hipSuccess);
//     ASSERT_EQ(hipMemcpy(&h_val, d_ptr, sizeof(int), hipMemcpyDeviceToHost), hipSuccess);
//     ASSERT_EQ(hipFree(d_ptr), hipSuccess);

//     ASSERT_EQ(h_val, 42);
// }


// cudaGetDeviceProperties_v2
TEST(cudaRT, GetDevicePropertiesV2) {
    int device;
    hipDeviceProp_t prop;

    ASSERT_EQ(hipGetDevice(&device), hipSuccess);
    ASSERT_EQ(hipGetDeviceProperties(&prop, device), hipSuccess);
    ASSERT_GT(prop.totalGlobalMem, 0);
}


//hipMemset
TEST(cudaRT, Memset) {
    const size_t size = 10 * sizeof(int);
    int* d_ptr = nullptr;

    ASSERT_EQ(hipMalloc(&d_ptr, size), hipSuccess);
    ASSERT_EQ(hipMemset(d_ptr, 0xAB, size), hipSuccess);
    ASSERT_EQ(hipFree(d_ptr), hipSuccess);
}

//hipMemGetInfo
TEST(cudaRT, MemGetInfo) {
    size_t free_mem = 0;
    size_t total_mem = 0;

    ASSERT_EQ(hipMemGetInfo(&free_mem, &total_mem), hipSuccess);
    ASSERT_GT(total_mem, 0);
    ASSERT_GT(free_mem, 0);
}

//hipStreamDestroy
TEST(cudaRT, StreamDestroy) {
    hipStream_t stream;
    ASSERT_EQ(hipStreamCreate(&stream), hipSuccess);
    ASSERT_NE(stream, nullptr);
    ASSERT_EQ(hipStreamDestroy(stream), hipSuccess);
}

//hipEventRecord
TEST(cudaRT, EventRecord) {
    hipEvent_t event;
    ASSERT_EQ(hipEventCreate(&event), hipSuccess);
    ASSERT_EQ(hipEventRecord(event), hipSuccess);
    ASSERT_EQ(hipEventSynchronize(event), hipSuccess);
    ASSERT_EQ(hipEventDestroy(event), hipSuccess);
}


//hipHostMalloc
TEST(cudaRT, MallocHost) {
    void* host_ptr = nullptr;
    ASSERT_EQ(hipHostMalloc(&host_ptr, 1024), hipSuccess);
    ASSERT_NE(host_ptr, nullptr);
    ASSERT_EQ(hipHostFree(host_ptr), hipSuccess);
}

//hipEventSynchronize
TEST(cudaRT, EventSynchronize) {
    hipEvent_t event;
    ASSERT_EQ(hipEventCreate(&event), hipSuccess);
    ASSERT_EQ(hipEventRecord(event), hipSuccess);
    ASSERT_EQ(hipEventSynchronize(event), hipSuccess);
    ASSERT_EQ(hipEventDestroy(event), hipSuccess);
}


//hipMemcpyAsync
TEST(cudaRT, MemcpyAsync) {
    const int N = 16;
    int h_src[N], h_dst[N];
    for (int i = 0; i < N; ++i) h_src[i] = i;

    int* d_ptr = nullptr;
    ASSERT_EQ(hipMalloc(&d_ptr, N * sizeof(int)), hipSuccess);

    ASSERT_EQ(hipMemcpyAsync(d_ptr, h_src, N * sizeof(int), hipMemcpyHostToDevice), hipSuccess);
    ASSERT_EQ(hipMemcpyAsync(h_dst, d_ptr, N * sizeof(int), hipMemcpyDeviceToHost), hipSuccess);

    ASSERT_EQ(hipDeviceSynchronize(), hipSuccess);
    ASSERT_EQ(hipFree(d_ptr), hipSuccess);

    for (int i = 0; i < N; ++i) {
        ASSERT_EQ(h_dst[i], h_src[i]);
    }
}

//hipGetDevice
TEST(cudaRT, GetDevice) {
    int device = -1;
    ASSERT_EQ(hipGetDevice(&device), hipSuccess);
    ASSERT_GE(device, 0);
}


//hipStreamCreate
TEST(cudaRT, StreamCreate) {
    hipStream_t stream;
    ASSERT_EQ(hipStreamCreate(&stream), hipSuccess);
    ASSERT_NE(stream, nullptr);
    ASSERT_EQ(hipStreamDestroy(stream), hipSuccess);
}


//hipMemcpy
TEST(cudaRT, Memcpy) {
    const int N = 16;
    int h_src[N], h_dst[N];
    for (int i = 0; i < N; ++i) h_src[i] = i;

    int* d_ptr = nullptr;
    ASSERT_EQ(hipMalloc(&d_ptr, N * sizeof(int)), hipSuccess);

    ASSERT_EQ(hipMemcpy(d_ptr, h_src, N * sizeof(int), hipMemcpyHostToDevice), hipSuccess);
    ASSERT_EQ(hipMemcpy(h_dst, d_ptr, N * sizeof(int), hipMemcpyDeviceToHost), hipSuccess);

    ASSERT_EQ(hipFree(d_ptr), hipSuccess);

    for (int i = 0; i < N; ++i) {
        ASSERT_EQ(h_dst[i], h_src[i]);
    }
}


//hipHostFree
TEST(cudaRT, FreeHost) {
    void* host_ptr = nullptr;
    ASSERT_EQ(hipHostMalloc(&host_ptr, 1024), hipSuccess);
    ASSERT_NE(host_ptr, nullptr);
    ASSERT_EQ(hipHostFree(host_ptr), hipSuccess);
}


//hipEventCreate
TEST(cudaRT, EventCreate) {
    hipEvent_t event;
    ASSERT_EQ(hipEventCreate(&event), hipSuccess);
    ASSERT_NE(event, nullptr);
    ASSERT_EQ(hipEventDestroy(event), hipSuccess);
}


//hipGetDeviceCount
TEST(cudaRT, GetDeviceCount) {
    int count = 0;
    ASSERT_EQ(hipGetDeviceCount(&count), hipSuccess);
    ASSERT_GT(count, 0);
}


// hipMalloc
TEST(cudaRT, Malloc) {
    int* d_ptr = nullptr;
    ASSERT_EQ(hipMalloc(&d_ptr, 1024 * sizeof(int)), hipSuccess);
    ASSERT_NE(d_ptr, nullptr);
    ASSERT_EQ(hipFree(d_ptr), hipSuccess);
}


//hipPeekAtLastError
TEST(cudaRT, PeekAtLastError) {
    // Launch an invalid kernel (no implementation)
    void* invalid_ptr = nullptr;
    hipError_t err = hipMemcpy(invalid_ptr, invalid_ptr, 100, hipMemcpyDeviceToDevice);

    // Peek at the error (should not be hipSuccess)
    hipError_t peek = hipPeekAtLastError();
    ASSERT_NE(peek, hipSuccess);
}


//hipStreamSynchronize
TEST(cudaRT, StreamSynchronize) {
    hipStream_t stream;
    ASSERT_EQ(hipStreamCreate(&stream), hipSuccess);
    ASSERT_EQ(hipStreamSynchronize(stream), hipSuccess);
    ASSERT_EQ(hipStreamDestroy(stream), hipSuccess);
}

//hipEventDestroy
TEST(cudaRT, EventDestroy) {
    hipEvent_t event;
    ASSERT_EQ(hipEventCreate(&event), hipSuccess);
    ASSERT_EQ(hipEventDestroy(event), hipSuccess);
}


//hipSetDevice
TEST(cudaRT, SetDevice) {
    int device_count = 0;
    ASSERT_EQ(hipGetDeviceCount(&device_count), hipSuccess);
    ASSERT_GT(device_count, 0);

    // Set to device 0
    ASSERT_EQ(hipSetDevice(0), hipSuccess);

    int current_device = -1;
    ASSERT_EQ(hipGetDevice(&current_device), hipSuccess);
    ASSERT_EQ(current_device, 0);
}

//hipGetLastError
TEST(cudaRT, GetLastError) {
    // Clear any previous error
    hipError_t reset = hipGetLastError();
    (void)reset; // suppress unused warning

    // Intentionally trigger an error
    hipError_t err = hipFree(nullptr);
    ASSERT_NE(err, hipSuccess);

    // Now check that last error matches
    hipError_t last = hipGetLastError();
    ASSERT_EQ(last, err);
}


//hipLaunchKernel
__global__ void launchKernelCheck(int* out) {
    if (threadIdx.x == 0) *out = 123;
}

TEST(cudaRT, LaunchKernel) {
    int* d_out = nullptr;
    int h_out = 0;

    ASSERT_EQ(hipMalloc(&d_out, sizeof(int)), hipSuccess);

    void* args[] = { &d_out };
    dim3 grid(1), block(1);

    ASSERT_EQ(hipLaunchKernel((void*)launchKernelCheck, grid, block, args, 0, nullptr), hipSuccess);
    ASSERT_EQ(hipDeviceSynchronize(), hipSuccess);
    ASSERT_EQ(hipMemcpy(&h_out, d_out, sizeof(int), hipMemcpyDeviceToHost), hipSuccess);
    ASSERT_EQ(hipFree(d_out), hipSuccess);

    ASSERT_EQ(h_out, 123);
}


//hipGetErrorString
TEST(cudaRT, GetErrorString) {
    hipError_t err = hipFree(nullptr);  // expected failure
    const char* str = hipGetErrorString(err);

    ASSERT_NE(err, hipSuccess);
    ASSERT_NE(str, nullptr);
}

