#include "hip/hip_runtime.h"
#include <gtest/gtest.h>
#include <hip/hip_runtime.h>
#include <hipDNN.h>

#define CUDA_CHECK(err) ASSERT_EQ((err), hipSuccess) << "CUDA Error: " << hipGetErrorString(err)
#define CUDNN_CHECK(err) ASSERT_EQ((err), HIPDNN_STATUS_SUCCESS) << "CUDNN Error: " << hipdnnGetErrorString(err)

//hipdnnCreate
TEST(cuDNN, Create) {
    hipdnnHandle_t handle;
    CUDNN_CHECK(hipdnnCreate(&handle));
    CUDNN_CHECK(hipdnnDestroy(handle));
}


//hipdnnDestroy
TEST(cuDNN, Destroy) {
    hipdnnHandle_t handle;
    CUDNN_CHECK(hipdnnCreate(&handle));
    CUDNN_CHECK(hipdnnDestroy(handle));
}


//hipdnnCreateTensorDescriptor
TEST(cuDNN, CreateTensorDescriptor) {
    hipdnnTensorDescriptor_t desc;
    CUDNN_CHECK(hipdnnCreateTensorDescriptor(&desc));
    CUDNN_CHECK(hipdnnDestroyTensorDescriptor(desc));
}


//hipdnnDestroyTensorDescriptor
TEST(cuDNN, DestroyTensorDescriptor) {
    hipdnnTensorDescriptor_t desc;
    CUDNN_CHECK(hipdnnCreateTensorDescriptor(&desc));
    CUDNN_CHECK(hipdnnDestroyTensorDescriptor(desc));
}

//hipdnnCreateActivationDescriptor
TEST(cuDNN, CreateActivationDescriptor) {
    hipdnnActivationDescriptor_t desc;
    CUDNN_CHECK(hipdnnCreateActivationDescriptor(&desc));
    CUDNN_CHECK(hipdnnDestroyActivationDescriptor(desc));
}


//hipdnnDestroyActivationDescriptor
TEST(cuDNN, DestroyActivationDescriptor) {
    hipdnnActivationDescriptor_t desc;
    CUDNN_CHECK(hipdnnCreateActivationDescriptor(&desc));
    CUDNN_CHECK(hipdnnDestroyActivationDescriptor(desc));
}

//hipdnnCreateConvolutionDescriptor
TEST(cuDNN, CreateConvolutionDescriptor) {
    hipdnnConvolutionDescriptor_t desc;
    CUDNN_CHECK(hipdnnCreateConvolutionDescriptor(&desc));
    CUDNN_CHECK(hipdnnDestroyConvolutionDescriptor(desc));
}

//hipdnnDestroyConvolutionDescriptor
TEST(cuDNN, DestroyConvolutionDescriptor) {
    hipdnnConvolutionDescriptor_t desc;
    CUDNN_CHECK(hipdnnCreateConvolutionDescriptor(&desc));
    CUDNN_CHECK(hipdnnDestroyConvolutionDescriptor(desc));
}


//hipdnnCreateFilterDescriptor
TEST(cuDNN, CreateFilterDescriptor) {
    hipdnnFilterDescriptor_t desc;
    CUDNN_CHECK(hipdnnCreateFilterDescriptor(&desc));
    CUDNN_CHECK(hipdnnDestroyFilterDescriptor(desc));
}


//hipdnnDestroyFilterDescriptor
TEST(cuDNN, DestroyFilterDescriptor) {
    hipdnnFilterDescriptor_t desc;
    CUDNN_CHECK(hipdnnCreateFilterDescriptor(&desc));
    CUDNN_CHECK(hipdnnDestroyFilterDescriptor(desc));
}


//hipdnnCreatePoolingDescriptor
TEST(cuDNN, CreatePoolingDescriptor) {
    hipdnnPoolingDescriptor_t desc;
    CUDNN_CHECK(hipdnnCreatePoolingDescriptor(&desc));
    CUDNN_CHECK(hipdnnDestroyPoolingDescriptor(desc));
}

//hipdnnDestroyPoolingDescriptor
TEST(cuDNN, DestroyPoolingDescriptor) {
    hipdnnPoolingDescriptor_t desc;
    CUDNN_CHECK(hipdnnCreatePoolingDescriptor(&desc));
    CUDNN_CHECK(hipdnnDestroyPoolingDescriptor(desc));
}


//hipdnnCreateLRNDescriptor
TEST(cuDNN, CreateLRNDescriptor) {
    hipdnnLRNDescriptor_t desc;
    CUDNN_CHECK(hipdnnCreateLRNDescriptor(&desc));
    CUDNN_CHECK(hipdnnDestroyLRNDescriptor(desc));
}


//hipdnnDestroyLRNDescriptor
TEST(cuDNN, DestroyLRNDescriptor) {
    hipdnnLRNDescriptor_t desc;
    CUDNN_CHECK(hipdnnCreateLRNDescriptor(&desc));
    CUDNN_CHECK(hipdnnDestroyLRNDescriptor(desc));
}

//hipdnnSetTensor4dDescriptorEx
TEST(cuDNN, SetTensor4dDescriptorEx) {
    hipdnnHandle_t handle;
    hipdnnTensorDescriptor_t desc;

    CUDNN_CHECK(hipdnnCreate(&handle));
    CUDNN_CHECK(hipdnnCreateTensorDescriptor(&desc));

    // Tensor dims: NCHW = 1x1x2x2
    int n = 1, c = 1, h = 2, w = 2;
    int nStride = c * h * w;     // 4
    int cStride = h * w;         // 4
    int hStride = w;             // 2
    int wStride = 1;

    CUDNN_CHECK(hipdnnSetTensor4dDescriptorEx(
        desc,
        HIPDNN_DATA_FLOAT,
        n, c, h, w,
        nStride, cStride, hStride, wStride));

    CUDNN_CHECK(hipdnnDestroyTensorDescriptor(desc));
    CUDNN_CHECK(hipdnnDestroy(handle));
}



//hipdnnSetActivationDescriptor
TEST(cuDNN, SetActivationDescriptor) {
    hipdnnActivationDescriptor_t desc;
    CUDNN_CHECK(hipdnnCreateActivationDescriptor(&desc));
    CUDNN_CHECK(hipdnnSetActivationDescriptor(desc, HIPDNN_ACTIVATION_RELU, HIPDNN_PROPAGATE_NAN, 0.0));
    CUDNN_CHECK(hipdnnDestroyActivationDescriptor(desc));
}

//hipdnnSetConvolution2dDescriptor
TEST(cuDNN, SetConvolution2dDescriptor) {
    hipdnnConvolutionDescriptor_t desc;
    CUDNN_CHECK(hipdnnCreateConvolutionDescriptor(&desc));
    CUDNN_CHECK(hipdnnSetConvolution2dDescriptor(
        desc, 1, 1, 1, 1, 1, 1, HIPDNN_CROSS_CORRELATION, HIPDNN_DATA_FLOAT));
    CUDNN_CHECK(hipdnnDestroyConvolutionDescriptor(desc));
}


//hipdnnSetFilter4dDescriptor
TEST(cuDNN, SetFilter4dDescriptor) {
    hipdnnFilterDescriptor_t desc;
    CUDNN_CHECK(hipdnnCreateFilterDescriptor(&desc));
    CUDNN_CHECK(hipdnnSetFilter4dDescriptor(
        desc, HIPDNN_DATA_FLOAT, HIPDNN_TENSOR_NCHW, 1, 1, 3, 3));
    CUDNN_CHECK(hipdnnDestroyFilterDescriptor(desc));
}


//hipdnnSetLRNDescriptor
TEST(cuDNN, SetLRNDescriptor) {
    hipdnnLRNDescriptor_t desc;
    CUDNN_CHECK(hipdnnCreateLRNDescriptor(&desc));
    CUDNN_CHECK(hipdnnSetLRNDescriptor(desc, 5, 1.0, 0.75, 1.0));
    CUDNN_CHECK(hipdnnDestroyLRNDescriptor(desc));
}

//hipdnnSetPooling2dDescriptor
TEST(cuDNN, SetPooling2dDescriptor) {
    hipdnnPoolingDescriptor_t desc;
    CUDNN_CHECK(hipdnnCreatePoolingDescriptor(&desc));
    CUDNN_CHECK(hipdnnSetPooling2dDescriptor(
        desc, HIPDNN_POOLING_MAX, HIPDNN_PROPAGATE_NAN,
        2, 2, 0, 0, 2, 2));
    CUDNN_CHECK(hipdnnDestroyPoolingDescriptor(desc));
}


//hipdnnSetStream
TEST(cuDNN, SetStream) {
    hipdnnHandle_t handle;
    hipStream_t stream;
    CUDNN_CHECK(hipdnnCreate(&handle));
    CUDA_CHECK(hipStreamCreate(&stream));

    CUDNN_CHECK(hipdnnSetStream(handle, stream));

    CUDA_CHECK(hipStreamDestroy(stream));
    CUDNN_CHECK(hipdnnDestroy(handle));
}

//hipdnnPoolingForward
TEST(cuDNN, PoolingForward) {
    hipdnnHandle_t handle;
    CUDNN_CHECK(hipdnnCreate(&handle));

    hipdnnTensorDescriptor_t inDesc, outDesc;
    hipdnnPoolingDescriptor_t poolDesc;

    float alpha = 1.0f, beta = 0.0f;
    int n = 1, c = 1, h = 4, w = 4;
    int poolH = 2, poolW = 2;

    float input[16] = {
        1, 2, 3, 4,
        5, 6, 7, 8,
        9,10,11,12,
       13,14,15,16
    };
    float output[4] = {0};

    float *d_in, *d_out;
    CUDA_CHECK(hipMalloc(&d_in, sizeof(input)));
    CUDA_CHECK(hipMalloc(&d_out, sizeof(output)));
    CUDA_CHECK(hipMemcpy(d_in, input, sizeof(input), hipMemcpyHostToDevice));

    CUDNN_CHECK(hipdnnCreateTensorDescriptor(&inDesc));
    CUDNN_CHECK(hipdnnCreateTensorDescriptor(&outDesc));
    CUDNN_CHECK(hipdnnCreatePoolingDescriptor(&poolDesc));

    CUDNN_CHECK(hipdnnSetTensor4dDescriptor(inDesc, HIPDNN_TENSOR_NCHW, HIPDNN_DATA_FLOAT, n, c, h, w));
    CUDNN_CHECK(hipdnnSetTensor4dDescriptor(outDesc, HIPDNN_TENSOR_NCHW, HIPDNN_DATA_FLOAT, n, c, 2, 2));
    CUDNN_CHECK(hipdnnSetPooling2dDescriptor(poolDesc, HIPDNN_POOLING_MAX, HIPDNN_PROPAGATE_NAN,
                                            poolH, poolW, 0, 0, poolH, poolW));

    CUDNN_CHECK(hipdnnPoolingForward(handle, poolDesc, &alpha, inDesc, d_in, &beta, outDesc, d_out));

    CUDA_CHECK(hipMemcpy(output, d_out, sizeof(output), hipMemcpyDeviceToHost));
    EXPECT_FLOAT_EQ(output[0], 6.0f);  // max(1,2,5,6)
    EXPECT_FLOAT_EQ(output[1], 8.0f);  // max(3,4,7,8)
    EXPECT_FLOAT_EQ(output[2], 14.0f); // max(9,10,13,14)
    EXPECT_FLOAT_EQ(output[3], 16.0f); // max(11,12,15,16)

    CUDA_CHECK(hipFree(d_in));
    CUDA_CHECK(hipFree(d_out));
    CUDNN_CHECK(hipdnnDestroyTensorDescriptor(inDesc));
    CUDNN_CHECK(hipdnnDestroyTensorDescriptor(outDesc));
    CUDNN_CHECK(hipdnnDestroyPoolingDescriptor(poolDesc));
    CUDNN_CHECK(hipdnnDestroy(handle));
}


//hipdnnSoftmaxForward
TEST(cuDNN, SoftmaxForward) {
    hipdnnHandle_t handle;
    CUDNN_CHECK(hipdnnCreate(&handle));

    hipdnnTensorDescriptor_t tensorDesc;
    float alpha = 1.0f, beta = 0.0f;
    float input[] = {1.0f, 2.0f, 3.0f};
    float output[3] = {0};

    float *d_in, *d_out;
    CUDA_CHECK(hipMalloc(&d_in, sizeof(input)));
    CUDA_CHECK(hipMalloc(&d_out, sizeof(output)));
    CUDA_CHECK(hipMemcpy(d_in, input, sizeof(input), hipMemcpyHostToDevice));

    CUDNN_CHECK(hipdnnCreateTensorDescriptor(&tensorDesc));
    CUDNN_CHECK(hipdnnSetTensor4dDescriptor(tensorDesc, HIPDNN_TENSOR_NCHW, HIPDNN_DATA_FLOAT, 1, 1, 1, 3));

    CUDNN_CHECK(hipdnnSoftmaxForward(handle, HIPDNN_SOFTMAX_ACCURATE, HIPDNN_SOFTMAX_MODE_INSTANCE,
                                    &alpha, tensorDesc, d_in, &beta, tensorDesc, d_out));

    CUDA_CHECK(hipMemcpy(output, d_out, sizeof(output), hipMemcpyDeviceToHost));
    float sum = output[0] + output[1] + output[2];
    EXPECT_NEAR(sum, 1.0f, 1e-5);  // Softmax should sum to 1

    CUDA_CHECK(hipFree(d_in));
    CUDA_CHECK(hipFree(d_out));
    CUDNN_CHECK(hipdnnDestroyTensorDescriptor(tensorDesc));
    CUDNN_CHECK(hipdnnDestroy(handle));
}

//hipdnnActivationForward
TEST(cuDNN, ActivationForward) {
    hipdnnHandle_t handle;
    CUDNN_CHECK(hipdnnCreate(&handle));

    hipdnnTensorDescriptor_t desc;
    hipdnnActivationDescriptor_t act;
    float alpha = 1.0f, beta = 0.0f;
    float input[] = {-1.0f, 0.0f, 1.0f};
    float output[3];

    float *d_in, *d_out;
    CUDA_CHECK(hipMalloc(&d_in, sizeof(input)));
    CUDA_CHECK(hipMalloc(&d_out, sizeof(output)));
    CUDA_CHECK(hipMemcpy(d_in, input, sizeof(input), hipMemcpyHostToDevice));

    CUDNN_CHECK(hipdnnCreateTensorDescriptor(&desc));
    CUDNN_CHECK(hipdnnSetTensor4dDescriptor(desc, HIPDNN_TENSOR_NCHW, HIPDNN_DATA_FLOAT, 1, 1, 1, 3));
    CUDNN_CHECK(hipdnnCreateActivationDescriptor(&act));
    CUDNN_CHECK(hipdnnSetActivationDescriptor(act, HIPDNN_ACTIVATION_RELU, HIPDNN_PROPAGATE_NAN, 0.0));

    CUDNN_CHECK(hipdnnActivationForward(handle, act, &alpha, desc, d_in, &beta, desc, d_out));

    CUDA_CHECK(hipMemcpy(output, d_out, sizeof(output), hipMemcpyDeviceToHost));
    EXPECT_FLOAT_EQ(output[0], 0.0f);
    EXPECT_FLOAT_EQ(output[1], 0.0f);
    EXPECT_FLOAT_EQ(output[2], 1.0f);

    CUDA_CHECK(hipFree(d_in));
    CUDA_CHECK(hipFree(d_out));
    CUDNN_CHECK(hipdnnDestroyActivationDescriptor(act));
    CUDNN_CHECK(hipdnnDestroyTensorDescriptor(desc));
    CUDNN_CHECK(hipdnnDestroy(handle));
}


//hipdnnActivationBackward
TEST(cuDNN, ActivationBackward) {
    hipdnnHandle_t handle;
    CUDNN_CHECK(hipdnnCreate(&handle));

    hipdnnTensorDescriptor_t desc;
    hipdnnActivationDescriptor_t act;
    float alpha = 1.0f, beta = 0.0f;

    float x[] = {-1.0f, 0.0f, 1.0f};
    float y[] = {0.0f, 0.0f, 1.0f}; // result of ReLU
    float dy[] = {1.0f, 1.0f, 1.0f}; // gradient from next layer
    float dx[3];

    float *d_x, *d_y, *d_dy, *d_dx;
    CUDA_CHECK(hipMalloc(&d_x, sizeof(x)));
    CUDA_CHECK(hipMalloc(&d_y, sizeof(y)));
    CUDA_CHECK(hipMalloc(&d_dy, sizeof(dy)));
    CUDA_CHECK(hipMalloc(&d_dx, sizeof(dx)));

    CUDA_CHECK(hipMemcpy(d_x, x, sizeof(x), hipMemcpyHostToDevice));
    CUDA_CHECK(hipMemcpy(d_y, y, sizeof(y), hipMemcpyHostToDevice));
    CUDA_CHECK(hipMemcpy(d_dy, dy, sizeof(dy), hipMemcpyHostToDevice));

    CUDNN_CHECK(hipdnnCreateTensorDescriptor(&desc));
    CUDNN_CHECK(hipdnnSetTensor4dDescriptor(desc, HIPDNN_TENSOR_NCHW, HIPDNN_DATA_FLOAT, 1, 1, 1, 3));
    CUDNN_CHECK(hipdnnCreateActivationDescriptor(&act));
    CUDNN_CHECK(hipdnnSetActivationDescriptor(act, HIPDNN_ACTIVATION_RELU, HIPDNN_PROPAGATE_NAN, 0.0));

    CUDNN_CHECK(hipdnnActivationBackward(handle, act, &alpha, desc, d_y, desc, d_dy, desc, d_x, &beta, desc, d_dx));

    CUDA_CHECK(hipMemcpy(dx, d_dx, sizeof(dx), hipMemcpyDeviceToHost));
    EXPECT_FLOAT_EQ(dx[0], 0.0f); // ReLU gradient = 0 if x < 0
    EXPECT_FLOAT_EQ(dx[1], 0.0f); // ReLU gradient = 0 if x = 0
    EXPECT_FLOAT_EQ(dx[2], 1.0f); // ReLU gradient = 1 if x > 0

    CUDA_CHECK(hipFree(d_x));
    CUDA_CHECK(hipFree(d_y));
    CUDA_CHECK(hipFree(d_dy));
    CUDA_CHECK(hipFree(d_dx));
    CUDNN_CHECK(hipdnnDestroyActivationDescriptor(act));
    CUDNN_CHECK(hipdnnDestroyTensorDescriptor(desc));
    CUDNN_CHECK(hipdnnDestroy(handle));
}


//hipdnnPoolingBackward
TEST(cuDNN, PoolingBackward) {
    hipdnnHandle_t handle;
    CUDNN_CHECK(hipdnnCreate(&handle));

    hipdnnTensorDescriptor_t inDesc, outDesc;
    hipdnnPoolingDescriptor_t poolDesc;

    float alpha = 1.0f, beta = 0.0f;
    float x[] = {1.0f, 5.0f, 3.0f, 4.0f};  // 2x2
    float y[] = {5.0f};                   // max pooled
    float dy[] = {1.0f};                  // gradient from next layer
    float dx[4];

    float *d_x, *d_y, *d_dy, *d_dx;
    CUDA_CHECK(hipMalloc(&d_x, sizeof(x)));
    CUDA_CHECK(hipMalloc(&d_y, sizeof(y)));
    CUDA_CHECK(hipMalloc(&d_dy, sizeof(dy)));
    CUDA_CHECK(hipMalloc(&d_dx, sizeof(dx)));

    CUDA_CHECK(hipMemcpy(d_x, x, sizeof(x), hipMemcpyHostToDevice));
    CUDA_CHECK(hipMemcpy(d_y, y, sizeof(y), hipMemcpyHostToDevice));
    CUDA_CHECK(hipMemcpy(d_dy, dy, sizeof(dy), hipMemcpyHostToDevice));

    CUDNN_CHECK(hipdnnCreateTensorDescriptor(&inDesc));
    CUDNN_CHECK(hipdnnCreateTensorDescriptor(&outDesc));
    CUDNN_CHECK(hipdnnCreatePoolingDescriptor(&poolDesc));

    CUDNN_CHECK(hipdnnSetTensor4dDescriptor(inDesc, HIPDNN_TENSOR_NCHW, HIPDNN_DATA_FLOAT, 1, 1, 2, 2));
    CUDNN_CHECK(hipdnnSetTensor4dDescriptor(outDesc, HIPDNN_TENSOR_NCHW, HIPDNN_DATA_FLOAT, 1, 1, 1, 1));
    CUDNN_CHECK(hipdnnSetPooling2dDescriptor(poolDesc, HIPDNN_POOLING_MAX, HIPDNN_PROPAGATE_NAN,
                                            2, 2, 0, 0, 2, 2));

    CUDNN_CHECK(hipdnnPoolingBackward(handle, poolDesc, &alpha,
                                     outDesc, d_y, outDesc, d_dy,
                                     inDesc, d_x, &beta,
                                     inDesc, d_dx));

    CUDA_CHECK(hipMemcpy(dx, d_dx, sizeof(dx), hipMemcpyDeviceToHost));
    EXPECT_FLOAT_EQ(dx[0], 0.0f);
    EXPECT_FLOAT_EQ(dx[1], 1.0f); // max was at position 1
    EXPECT_FLOAT_EQ(dx[2], 0.0f);
    EXPECT_FLOAT_EQ(dx[3], 0.0f);

    CUDA_CHECK(hipFree(d_x));
    CUDA_CHECK(hipFree(d_y));
    CUDA_CHECK(hipFree(d_dy));
    CUDA_CHECK(hipFree(d_dx));
    CUDNN_CHECK(hipdnnDestroyTensorDescriptor(inDesc));
    CUDNN_CHECK(hipdnnDestroyTensorDescriptor(outDesc));
    CUDNN_CHECK(hipdnnDestroyPoolingDescriptor(poolDesc));
    CUDNN_CHECK(hipdnnDestroy(handle));
}


//hipdnnSoftmaxBackward
TEST(cuDNN, SoftmaxBackward) {
    hipdnnHandle_t handle;
    CUDNN_CHECK(hipdnnCreate(&handle));

    hipdnnTensorDescriptor_t tensorDesc;
    float alpha = 1.0f, beta = 0.0f;

    float y[]  = {0.1f, 0.6f, 0.3f};
    float dy[] = {1.0f, 1.0f, 1.0f};
    float dx[3] = {0};

    float *d_y, *d_dy, *d_dx;
    CUDA_CHECK(hipMalloc(&d_y, sizeof(y)));
    CUDA_CHECK(hipMalloc(&d_dy, sizeof(dy)));
    CUDA_CHECK(hipMalloc(&d_dx, sizeof(dx)));

    CUDA_CHECK(hipMemcpy(d_y, y, sizeof(y), hipMemcpyHostToDevice));
    CUDA_CHECK(hipMemcpy(d_dy, dy, sizeof(dy), hipMemcpyHostToDevice));

    CUDNN_CHECK(hipdnnCreateTensorDescriptor(&tensorDesc));
    // NCHW layout: 1 sample, 1 channel, 1x3 spatial
    CUDNN_CHECK(hipdnnSetTensor4dDescriptor(
        tensorDesc, HIPDNN_TENSOR_NCHW, HIPDNN_DATA_FLOAT, 1, 1, 1, 3));

    CUDNN_CHECK(hipdnnSoftmaxBackward(
        handle,
        HIPDNN_SOFTMAX_ACCURATE,
        HIPDNN_SOFTMAX_MODE_INSTANCE,
        &alpha,
        tensorDesc, d_y,
        tensorDesc, d_dy,
        &beta,
        tensorDesc, d_dx));

    CUDA_CHECK(hipFree(d_y));
    CUDA_CHECK(hipFree(d_dy));
    CUDA_CHECK(hipFree(d_dx));
    CUDNN_CHECK(hipdnnDestroyTensorDescriptor(tensorDesc));
    CUDNN_CHECK(hipdnnDestroy(handle));
}


