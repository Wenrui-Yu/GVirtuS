#include "hip/hip_runtime.h"
#include <gtest/gtest.h>
#include <hip/hip_runtime.h>
#include <hipDNN.h>

#define CUDA_CHECK(err) ASSERT_EQ((err), hipSuccess)
#define CUDNN_CHECK(err) ASSERT_EQ((err), HIPDNN_STATUS_SUCCESS)

TEST(cuDNN, CreateDestroy) {
    hipdnnHandle_t handle;
    CUDNN_CHECK(hipdnnCreate(&handle));
    CUDNN_CHECK(hipdnnDestroy(handle));
}

TEST(cuDNN, GetVersion) {
    size_t version = hipdnnGetVersion();
    ASSERT_GT(version, 0);
    std::cout << "cuDNN version: " << version << std::endl;
}

TEST(cuDNN, SetGetStream) {
    hipdnnHandle_t handle;
    CUDNN_CHECK(hipdnnCreate(&handle));

    hipStream_t stream;
    CUDA_CHECK(hipStreamCreate(&stream));

    CUDNN_CHECK(hipdnnSetStream(handle, stream));

    hipStream_t got_stream;
    CUDNN_CHECK(hipdnnGetStream(handle, &got_stream));
    ASSERT_EQ(stream, got_stream);

    CUDA_CHECK(hipStreamDestroy(stream));
    CUDNN_CHECK(hipdnnDestroy(handle));
}

TEST(cuDNN, TensorDescriptorCreateDestroy) {
    hipdnnTensorDescriptor_t desc;
    CUDNN_CHECK(hipdnnCreateTensorDescriptor(&desc));
    CUDNN_CHECK(hipdnnDestroyTensorDescriptor(desc));
}

TEST(cuDNN, TensorDescriptorSetGet) {
    hipdnnTensorDescriptor_t desc;
    CUDNN_CHECK(hipdnnCreateTensorDescriptor(&desc));

    CUDNN_CHECK(hipdnnSetTensor4dDescriptor(desc, HIPDNN_TENSOR_NCHW, HIPDNN_DATA_FLOAT, 1, 2, 3, 4));

    hipdnnDataType_t dataType;
    int n, c, h, w, nStride, cStride, hStride, wStride;

    CUDNN_CHECK(hipdnnGetTensor4dDescriptor(desc, &dataType, &n, &c, &h, &w, &nStride, &cStride, &hStride, &wStride));
    ASSERT_EQ(n, 1);
    ASSERT_EQ(c, 2);
    ASSERT_EQ(h, 3);
    ASSERT_EQ(w, 4);

    CUDNN_CHECK(hipdnnDestroyTensorDescriptor(desc));
}

TEST(cuDNN, ActivationForwardReLU) {
    hipdnnHandle_t handle;
    CUDNN_CHECK(hipdnnCreate(&handle));

    hipdnnTensorDescriptor_t desc;
    CUDNN_CHECK(hipdnnCreateTensorDescriptor(&desc));
    CUDNN_CHECK(hipdnnSetTensor4dDescriptor(desc, HIPDNN_TENSOR_NCHW, HIPDNN_DATA_FLOAT, 1, 1, 1, 3));

    hipdnnActivationDescriptor_t actDesc;
    CUDNN_CHECK(hipdnnCreateActivationDescriptor(&actDesc));
    CUDNN_CHECK(hipdnnSetActivationDescriptor(actDesc, HIPDNN_ACTIVATION_RELU, HIPDNN_PROPAGATE_NAN, 0.0));

    float h_input[] = {-1.0f, 0.0f, 2.0f};
    float h_output[3] = {};

    float *d_input, *d_output;
    CUDA_CHECK(hipMalloc(&d_input, sizeof(h_input)));
    CUDA_CHECK(hipMalloc(&d_output, sizeof(h_output)));
    CUDA_CHECK(hipMemcpy(d_input, h_input, sizeof(h_input), hipMemcpyHostToDevice));

    float alpha = 1.0f, beta = 0.0f;
    CUDNN_CHECK(hipdnnActivationForward(handle, actDesc, &alpha, desc, d_input, &beta, desc, d_output));
    CUDA_CHECK(hipMemcpy(h_output, d_output, sizeof(h_output), hipMemcpyDeviceToHost));

    ASSERT_FLOAT_EQ(h_output[0], 0.0f);
    ASSERT_FLOAT_EQ(h_output[1], 0.0f);
    ASSERT_FLOAT_EQ(h_output[2], 2.0f);

    CUDA_CHECK(hipFree(d_input));
    CUDA_CHECK(hipFree(d_output));
    CUDNN_CHECK(hipdnnDestroyTensorDescriptor(desc));
    CUDNN_CHECK(hipdnnDestroyActivationDescriptor(actDesc));
    CUDNN_CHECK(hipdnnDestroy(handle));
}

TEST(cuDNN, PoolingForwardMax) {
    hipdnnHandle_t handle;
    CUDNN_CHECK(hipdnnCreate(&handle));

    hipdnnTensorDescriptor_t in_desc, out_desc;
    hipdnnPoolingDescriptor_t pool_desc;

    CUDNN_CHECK(hipdnnCreateTensorDescriptor(&in_desc));
    CUDNN_CHECK(hipdnnCreateTensorDescriptor(&out_desc));
    CUDNN_CHECK(hipdnnCreatePoolingDescriptor(&pool_desc));

    CUDNN_CHECK(hipdnnSetTensor4dDescriptor(in_desc, HIPDNN_TENSOR_NCHW, HIPDNN_DATA_FLOAT, 1, 1, 2, 2));
    CUDNN_CHECK(hipdnnSetTensor4dDescriptor(out_desc, HIPDNN_TENSOR_NCHW, HIPDNN_DATA_FLOAT, 1, 1, 1, 1));
    CUDNN_CHECK(hipdnnSetPooling2dDescriptor(pool_desc, HIPDNN_POOLING_MAX, HIPDNN_PROPAGATE_NAN,
                                            2, 2, 0, 0, 1, 1));

    float h_input[] = {1, 2, 3, 4}; // max is 4
    float h_output[1] = {};

    float *d_input, *d_output;
    CUDA_CHECK(hipMalloc(&d_input, sizeof(h_input)));
    CUDA_CHECK(hipMalloc(&d_output, sizeof(h_output)));
    CUDA_CHECK(hipMemcpy(d_input, h_input, sizeof(h_input), hipMemcpyHostToDevice));

    float alpha = 1.0f, beta = 0.0f;
    CUDNN_CHECK(hipdnnPoolingForward(handle, pool_desc, &alpha, in_desc, d_input, &beta, out_desc, d_output));
    CUDA_CHECK(hipMemcpy(h_output, d_output, sizeof(h_output), hipMemcpyDeviceToHost));

    ASSERT_FLOAT_EQ(h_output[0], 4.0f);

    CUDA_CHECK(hipFree(d_input));
    CUDA_CHECK(hipFree(d_output));
    CUDNN_CHECK(hipdnnDestroyTensorDescriptor(in_desc));
    CUDNN_CHECK(hipdnnDestroyTensorDescriptor(out_desc));
    CUDNN_CHECK(hipdnnDestroyPoolingDescriptor(pool_desc));
    CUDNN_CHECK(hipdnnDestroy(handle));
}

TEST(cuDNN, FilterDescriptorCreateDestroy) {
    hipdnnFilterDescriptor_t filterDesc;
    CUDNN_CHECK(hipdnnCreateFilterDescriptor(&filterDesc));
    CUDNN_CHECK(hipdnnDestroyFilterDescriptor(filterDesc));
}

TEST(cuDNN, ErrorString) {
    const char* msg = hipdnnGetErrorString(HIPDNN_STATUS_ALLOC_FAILED);
    ASSERT_TRUE(msg != nullptr);
}
