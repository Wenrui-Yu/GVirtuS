#include <gtest/gtest.h>
#include <iostream>
#include <hiprand.h>

#define CUDA_CHECK(err) ASSERT_EQ((err), hipSuccess) << "CUDA error: " << hipGetErrorString(err)
#define CURAND_CHECK(err) ASSERT_EQ((err), HIPRAND_STATUS_SUCCESS)

//hiprandCreateGenerator
TEST(cuRAND, CreateGenerator) {
    hiprandGenerator_t gen;
    CURAND_CHECK(hiprandCreateGenerator(&gen, HIPRAND_RNG_PSEUDO_DEFAULT));
    CURAND_CHECK(hiprandDestroyGenerator(gen));
}


//hiprandDestroyGenerator
TEST(cuRAND, DestroyGenerator) {
    hiprandGenerator_t gen;
    CURAND_CHECK(hiprandCreateGenerator(&gen, HIPRAND_RNG_PSEUDO_DEFAULT));
    CURAND_CHECK(hiprandDestroyGenerator(gen));
}


//hiprandGenerate
TEST(cuRAND, Generate) {
    hiprandGenerator_t gen;
    CURAND_CHECK(hiprandCreateGenerator(&gen, HIPRAND_RNG_PSEUDO_DEFAULT));
    CURAND_CHECK(hiprandSetPseudoRandomGeneratorSeed(gen, 1234ULL));

    const size_t n = 10;
    unsigned int* d_data;
    CUDA_CHECK(hipMalloc(&d_data, n * sizeof(unsigned int)));

    CURAND_CHECK(hiprandGenerate(gen, d_data, n));

    CUDA_CHECK(hipFree(d_data));
    CURAND_CHECK(hiprandDestroyGenerator(gen));
}

//hiprandGenerateNormal
TEST(cuRAND, GenerateNormal) {
    hiprandGenerator_t gen;
    CURAND_CHECK(hiprandCreateGenerator(&gen, HIPRAND_RNG_PSEUDO_DEFAULT));
    CURAND_CHECK(hiprandSetPseudoRandomGeneratorSeed(gen, 1234ULL));

    const size_t n = 10;
    float* d_data;
    CUDA_CHECK(hipMalloc(&d_data, n * sizeof(float)));

    CURAND_CHECK(hiprandGenerateNormal(gen, d_data, n, 0.0f, 1.0f));

    CUDA_CHECK(hipFree(d_data));
    CURAND_CHECK(hiprandDestroyGenerator(gen));
}


//hiprandGenerateNormalDouble
TEST(cuRAND, GenerateNormalDouble) {
    hiprandGenerator_t gen;
    CURAND_CHECK(hiprandCreateGenerator(&gen, HIPRAND_RNG_PSEUDO_DEFAULT));
    CURAND_CHECK(hiprandSetPseudoRandomGeneratorSeed(gen, 1234ULL));

    const size_t n = 10;
    double* d_data;
    CUDA_CHECK(hipMalloc(&d_data, n * sizeof(double)));

    CURAND_CHECK(hiprandGenerateNormalDouble(gen, d_data, n, 0.0, 1.0));

    CUDA_CHECK(hipFree(d_data));
    CURAND_CHECK(hiprandDestroyGenerator(gen));
}


//hiprandGenerateUniform
TEST(cuRAND, GenerateUniform) {
    hiprandGenerator_t gen;
    CURAND_CHECK(hiprandCreateGenerator(&gen, HIPRAND_RNG_PSEUDO_DEFAULT));
    CURAND_CHECK(hiprandSetPseudoRandomGeneratorSeed(gen, 1234ULL));

    const size_t n = 10;
    float* d_data;
    CUDA_CHECK(hipMalloc(&d_data, n * sizeof(float)));

    CURAND_CHECK(hiprandGenerateUniform(gen, d_data, n));

    CUDA_CHECK(hipFree(d_data));
    CURAND_CHECK(hiprandDestroyGenerator(gen));
}


//hiprandGenerateUniformDouble
TEST(cuRAND, GenerateUniformDouble) {
    hiprandGenerator_t gen;
    CURAND_CHECK(hiprandCreateGenerator(&gen, HIPRAND_RNG_PSEUDO_DEFAULT));
    CURAND_CHECK(hiprandSetPseudoRandomGeneratorSeed(gen, 1234ULL));

    const size_t n = 10;
    double* d_data;
    CUDA_CHECK(hipMalloc(&d_data, n * sizeof(double)));

    CURAND_CHECK(hiprandGenerateUniformDouble(gen, d_data, n));

    CUDA_CHECK(hipFree(d_data));
    CURAND_CHECK(hiprandDestroyGenerator(gen));
}


//hiprandSetPseudoRandomGeneratorSeed
TEST(cuRAND, SetPseudoRandomGeneratorSeed) {
    hiprandGenerator_t gen;
    CURAND_CHECK(hiprandCreateGenerator(&gen, HIPRAND_RNG_PSEUDO_DEFAULT));
    CURAND_CHECK(hiprandSetPseudoRandomGeneratorSeed(gen, 1234ULL));
    CURAND_CHECK(hiprandDestroyGenerator(gen));
}


//hiprandSetGeneratorOffset
TEST(cuRAND, SetGeneratorOffset) {
    hiprandGenerator_t gen1, gen2;

    // Create 2 identical generators
    CURAND_CHECK(hiprandCreateGenerator(&gen1, HIPRAND_RNG_PSEUDO_DEFAULT));
    CURAND_CHECK(hiprandCreateGenerator(&gen2, HIPRAND_RNG_PSEUDO_DEFAULT));

    // Set same seed
    CURAND_CHECK(hiprandSetPseudoRandomGeneratorSeed(gen1, 1234ULL));
    CURAND_CHECK(hiprandSetPseudoRandomGeneratorSeed(gen2, 1234ULL));

    // Set different offsets
    size_t offset = 1000;
    CURAND_CHECK(hiprandSetGeneratorOffset(gen1, offset));

    // Generate same count from both, but gen2 will generate offset + count
    const int count = 10;
    std::vector<float> out1(count), out2(offset + count);

    float *d_out1, *d_out2;
    hipMalloc(&d_out1, count * sizeof(float));
    hipMalloc(&d_out2, (offset + count) * sizeof(float));

    // Generate for both
    CURAND_CHECK(hiprandGenerateUniform(gen1, d_out1, count));
    CURAND_CHECK(hiprandGenerateUniform(gen2, d_out2, offset + count));

    // Copy and compare
    hipMemcpy(out1.data(), d_out1, count * sizeof(float), hipMemcpyDeviceToHost);
    hipMemcpy(out2.data(), d_out2, (offset + count) * sizeof(float), hipMemcpyDeviceToHost);

    // The output of gen1 should match gen2's values starting at [offset]
    for (int i = 0; i < count; ++i) {
        EXPECT_FLOAT_EQ(out1[i], out2[i + offset]);
    }

    CURAND_CHECK(hiprandDestroyGenerator(gen1));
    CURAND_CHECK(hiprandDestroyGenerator(gen2));
    hipFree(d_out1);
    hipFree(d_out2);
}