#include <gtest/gtest.h>
#include <iostream>
#include <hiprand.h>

#define CUDA_CHECK(err) ASSERT_EQ((err), hipSuccess) << "CUDA error: " << hipGetErrorString(err)
#define CURAND_CHECK(err) ASSERT_EQ((err), HIPRAND_STATUS_SUCCESS)

//hiprandCreateGenerator
TEST(cuRAND, CreateGenerator) {
    hiprandGenerator_t gen;
    CURAND_CHECK(hiprandCreateGenerator(&gen, HIPRAND_RNG_PSEUDO_DEFAULT));
    CURAND_CHECK(hiprandDestroyGenerator(gen));
}


//hiprandDestroyGenerator
TEST(cuRAND, DestroyGenerator) {
    hiprandGenerator_t gen;
    CURAND_CHECK(hiprandCreateGenerator(&gen, HIPRAND_RNG_PSEUDO_DEFAULT));
    CURAND_CHECK(hiprandDestroyGenerator(gen));
}


//hiprandGenerate
TEST(cuRAND, Generate) {
    hiprandGenerator_t gen;
    CURAND_CHECK(hiprandCreateGenerator(&gen, HIPRAND_RNG_PSEUDO_DEFAULT));
    CURAND_CHECK(hiprandSetPseudoRandomGeneratorSeed(gen, 1234ULL));

    const size_t n = 10;
    unsigned int* d_data;
    CUDA_CHECK(hipMalloc(&d_data, n * sizeof(unsigned int)));

    CURAND_CHECK(hiprandGenerate(gen, d_data, n));

    CUDA_CHECK(hipFree(d_data));
    CURAND_CHECK(hiprandDestroyGenerator(gen));
}

//hiprandGenerateNormal
TEST(cuRAND, GenerateNormal) {
    hiprandGenerator_t gen;
    CURAND_CHECK(hiprandCreateGenerator(&gen, HIPRAND_RNG_PSEUDO_DEFAULT));
    CURAND_CHECK(hiprandSetPseudoRandomGeneratorSeed(gen, 1234ULL));

    const size_t n = 10;
    float* d_data;
    CUDA_CHECK(hipMalloc(&d_data, n * sizeof(float)));

    CURAND_CHECK(hiprandGenerateNormal(gen, d_data, n, 0.0f, 1.0f));

    CUDA_CHECK(hipFree(d_data));
    CURAND_CHECK(hiprandDestroyGenerator(gen));
}


//hiprandGenerateNormalDouble
TEST(cuRAND, GenerateNormalDouble) {
    hiprandGenerator_t gen;
    CURAND_CHECK(hiprandCreateGenerator(&gen, HIPRAND_RNG_PSEUDO_DEFAULT));
    CURAND_CHECK(hiprandSetPseudoRandomGeneratorSeed(gen, 1234ULL));

    const size_t n = 10;
    double* d_data;
    CUDA_CHECK(hipMalloc(&d_data, n * sizeof(double)));

    CURAND_CHECK(hiprandGenerateNormalDouble(gen, d_data, n, 0.0, 1.0));

    CUDA_CHECK(hipFree(d_data));
    CURAND_CHECK(hiprandDestroyGenerator(gen));
}


//hiprandGenerateUniform
TEST(cuRAND, GenerateUniform) {
    hiprandGenerator_t gen;
    CURAND_CHECK(hiprandCreateGenerator(&gen, HIPRAND_RNG_PSEUDO_DEFAULT));
    CURAND_CHECK(hiprandSetPseudoRandomGeneratorSeed(gen, 1234ULL));

    const size_t n = 10;
    float* d_data;
    CUDA_CHECK(hipMalloc(&d_data, n * sizeof(float)));

    CURAND_CHECK(hiprandGenerateUniform(gen, d_data, n));

    CUDA_CHECK(hipFree(d_data));
    CURAND_CHECK(hiprandDestroyGenerator(gen));
}


//hiprandGenerateUniformDouble
TEST(cuRAND, GenerateUniformDouble) {
    hiprandGenerator_t gen;
    CURAND_CHECK(hiprandCreateGenerator(&gen, HIPRAND_RNG_PSEUDO_DEFAULT));
    CURAND_CHECK(hiprandSetPseudoRandomGeneratorSeed(gen, 1234ULL));

    const size_t n = 10;
    double* d_data;
    CUDA_CHECK(hipMalloc(&d_data, n * sizeof(double)));

    CURAND_CHECK(hiprandGenerateUniformDouble(gen, d_data, n));

    CUDA_CHECK(hipFree(d_data));
    CURAND_CHECK(hiprandDestroyGenerator(gen));
}


//hiprandSetPseudoRandomGeneratorSeed
TEST(cuRAND, SetPseudoRandomGeneratorSeed) {
    hiprandGenerator_t gen;
    CURAND_CHECK(hiprandCreateGenerator(&gen, HIPRAND_RNG_PSEUDO_DEFAULT));
    CURAND_CHECK(hiprandSetPseudoRandomGeneratorSeed(gen, 1234ULL));
    CURAND_CHECK(hiprandDestroyGenerator(gen));
}
